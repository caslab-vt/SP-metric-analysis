#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

#include "depthwise_convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"

__global__ void DepthwiseConv2dGPUKernelNCHW(
	const float* input,const int in_rows, const int in_cols, const int in_depth,
	const float* filter, const int filter_rows, const int filter_cols,
	const int stride,const int pad_rows,const int pad_cols,
	const int out_rows,const int out_cols,const int out_depth,
	float* output, int num_outputs,int offset) {

	int thread_id = blockIdx.x * blockDim.x + offset +  threadIdx.x;
	//int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (thread_id >= num_outputs) return;



	const int OC = thread_id % out_cols;//width
	const int OR = (thread_id / out_cols) % out_rows;//height
	const int OD = (thread_id / out_cols / out_rows) % out_depth;//channel
	const int OB = thread_id / out_cols / out_rows / out_depth;//batch size

	const int in_d = OD ;
	const int input_offset_temp = (OB * in_depth + in_d) * (in_rows * in_cols);
	const int input_row_start = OR * stride - pad_rows;
	const int input_col_start = OC * stride - pad_cols;
	const int input_row_end = input_row_start + filter_rows;
	const int input_col_end = input_col_start + filter_cols;

	float sum = 0;
	if (input_row_start >= 0 && input_col_start >= 0 &&
		input_row_end < in_rows && input_col_end < in_cols)
	{
		#pragma unroll
			for (int f_r = 0; f_r < filter_rows; ++f_r) {
				const int in_r = input_row_start + f_r;
				#pragma unroll
				for (int f_c = 0; f_c < filter_cols; ++f_c) {
					const int in_c = input_col_start + f_c;

					const int input_offset =
						(input_offset_temp)+(in_r * in_cols) + in_c;
					const int filter_offset =f_c + filter_cols * f_r +OD*filter_cols*filter_rows;
					sum += (*(input + input_offset)) * (*(filter + filter_offset));
				}
			}
		}
	else {
		#pragma unroll
		for (int f_r = 0; f_r < filter_rows; ++f_r) {
				const int in_r = input_row_start + f_r;
				#pragma unroll
				for (int f_c = 0; f_c < filter_cols; ++f_c) {
					const int in_c = input_col_start + f_c;

					if (in_r >= 0 && in_r < in_rows && in_c >= 0 && in_c < in_cols) {
						//const int in_c = input_col_start + f_c;

						const int input_offset =
							(input_offset_temp)+(in_r * in_cols) + in_c;

						const int filter_offset = f_c + filter_cols * f_r + OD*filter_cols*filter_rows;
						sum += (*(input + input_offset)) * (*(filter + filter_offset));
					}
				}
			}
		}

	output[thread_id] = sum;

}

__global__ void DepthwiseConv2dBackpropFilterGPUKernelNCHW(const float* out_backprop,
			const int stride, const int pad_rows, const int pad_cols, const int out_rows, const int out_cols, const int out_depth,
			const float* input, const int in_rows, const int in_cols, const int in_depth,
			float* filter_backprop, const int filter_rows, const int filter_cols,
			int num_out_backprop) {

	int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (thread_id >= num_out_backprop) return;


	const int out_c = thread_id % out_cols;
	const int out_r = (thread_id / out_cols) % out_rows;
	const int out_d = (thread_id / out_cols / out_rows) % out_depth;

	const int b = thread_id / out_depth / out_cols / out_rows;
	const int in_d = out_d;


	const int in_r_start = out_r * stride - pad_rows;
	const int in_c_start = out_c * stride - pad_cols;
	const int in_r_end = in_r_start + filter_rows;
	const int in_c_end = in_c_start + filter_cols;

	const int out_backprop_offset = (b * out_depth * out_rows * out_cols) +
				(out_d * out_rows * out_cols) +(out_r * out_cols) + (out_c);

	const float out_bp = *(out_backprop + out_backprop_offset);
	if (in_r_start >= 0 && in_c_start >= 0 && in_r_end < in_rows &&in_c_end < in_cols) {
		#pragma unroll 
		for (int f_r = 0; f_r < filter_rows; ++f_r) {
			const int in_r = in_r_start + f_r;
			const int input_offset_temp = (b * in_depth * in_rows * in_cols) +
						(in_d * in_rows * in_cols) +(in_r * in_cols);

			#pragma unroll 
			for (int f_c = 0; f_c < filter_cols; ++f_c) {
						const int in_c = in_c_start + f_c;
						const int input_offset = input_offset_temp + in_c;
						float partial_sum = (*(input + input_offset)) * out_bp;
						float* addr = filter_backprop + f_c + filter_cols * f_r + out_d*filter_cols*filter_rows;
						atomicAdd(addr, partial_sum);
					}
				}
			}
			else {
				#pragma unroll 
				for (int f_r = 0; f_r < filter_rows; ++f_r) {
					const int in_r = in_r_start + f_r;
					const int input_offset_temp = (b * in_depth * in_rows * in_cols) +(in_d * in_rows * in_cols) +(in_r * in_cols);
					#pragma unroll 
					for (int f_c = 0; f_c < filter_cols; ++f_c) {
						const int in_c = in_c_start + f_c;
						if (in_r >= 0 && in_r < in_rows && in_c >= 0 && in_c < in_cols) {
							const int input_offset = input_offset_temp + in_c;
							float partial_sum = (*(input + input_offset)) * out_bp;
							float* addr =filter_backprop + f_c + filter_cols * f_r + out_d*filter_cols*filter_rows;
							atomicAdd(addr, partial_sum);
						}
					}
				}

		}
	}



__global__ void DepthwiseConv2dBackpropInputGPUKernelNCHW(
		const float* out_backprop, const int out_rows, const int out_cols, const int out_depth,
		const float* filter, const int filter_rows, const int filter_cols,
		float* in_backprop, const int in_rows, const int in_cols, const int in_depth,
		const int stride, const int pad_rows, const int pad_cols,int num_in_backprop)
{
		int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
		if (thread_id >= num_in_backprop) return;

		const int in_c = thread_id % in_cols;
		const int in_r = (thread_id / in_cols) % in_rows;
		const int in_d = (thread_id / in_cols / in_rows) % in_depth;
		const int b = thread_id / in_depth / in_cols / in_rows;

		float sum = 0;


		const int out_r_start =max(0, (in_r - filter_rows + pad_rows + stride) / stride);
		const int out_r_end = min(out_rows - 1, (in_r + pad_rows) / stride);
		const int out_c_start =
			max(0, (in_c - filter_cols + pad_cols + stride) / stride);
		const int out_c_end = min(out_cols - 1, (in_c + pad_cols) / stride);


		#pragma unroll 
			for (int out_r = out_r_start; out_r <= out_r_end; ++out_r) {
				const int f_r = in_r + pad_rows - out_r * stride;


				for (int out_c = out_c_start; out_c <= out_c_end; ++out_c) {
					const int f_c = in_c + pad_cols - out_c * stride;
					const int filter_offset = f_c + filter_cols * f_r + in_d *filter_cols*filter_rows;

					const int out_backprop_offset =
						(b * out_depth * out_rows * out_cols) +
						(in_d * out_rows * out_cols) + (out_r * out_cols) + (out_c);

					sum += (*(out_backprop + out_backprop_offset)) *
						(*(filter + filter_offset));
				}
			}
		in_backprop[thread_id] = sum;

}


void forward_depthwise_convolutional_layer_gpu(depthwise_convolutional_layer l, network_state state)
{
	//cuda_pull_array(l.output_gpu, l.output, l.c*l.out_h*l.out_w);//add by hjimce for debug
    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);

	int size = l.out_h*l.out_w*l.batch*l.n;

	int num_blocks = get_number_of_blocks(size, BLOCK);
	int extra_blocks = num_blocks %4;
	int stream_id = 0;
	num_blocks = num_blocks /4;
	int loopcount = 0;
	int offset = 0;
	hipDeviceSynchronize();
	for (loopcount = 0; loopcount < num_blocks;loopcount++)
	{
		offset = loopcount * 4*BLOCK;
		stream_id = loopcount%4;
		DepthwiseConv2dGPUKernelNCHW <<<4, BLOCK,0,get_cuda_stream_32(stream_id) >>>(state.input,l.h,l.w,l.c,l.weights_gpu, l.size, l.size,l.stride, l.pad, l.pad,l.out_h, l.out_w, l.n,l.output_gpu, size,offset);	
	}
	hipDeviceSynchronize();
	if(extra_blocks)
	{
		stream_id = loopcount%4;
		offset = loopcount * 4*BLOCK;
		DepthwiseConv2dGPUKernelNCHW <<<extra_blocks, BLOCK,0,get_cuda_stream_32(stream_id) >>>(state.input,l.h,l.w,l.c,l.weights_gpu, l.size, l.size,l.stride, l.pad, l.pad,l.out_h, l.out_w, l.n,l.output_gpu, size,offset);
	}	
	


	/*DepthwiseConv2dGPUKernelNCHW << <4, BLOCK,0,get_cuda_stream_32(stream_id) >> >(state.input,l.h,l.w,l.c,l.weights_gpu, l.size, l.size,l.stride, l.pad, l.pad,l.out_h, l.out_w, l.n,l.output_gpu, size,offset);*/
	check_error(hipPeekAtLastError());
    /*int i;
    int k = l.size*l.size;
    int n = l.out_w*l.out_h;

	for (int b = 0; b < l.batch; ++b) {
		for (int c = 0; c<l.c; c++)
		{

			float *aoffset = l.weights_gpu + c*l.size*l.size;
			float *boffset = state.workspace;
			float *coffset = l.output_gpu + c*l.out_h*l.out_w + b*l.n*l.out_h*l.out_w;
			float *intput_offset = state.input + c*l.h*l.w + b*l.c*l.h*l.w;
			//im2col_gpu_ext(intput_offset, 1, l.h, l.w,
			//	l.size, l.stride, l.pad, boffset);

			 im2col_gpu_ext(intput_offset,          // input
                   	1,         // input channels
                    l.h, l.w,               // input size (h, w)
                    l.size, l.size,         // kernel size (h, w)
                    l.pad, l.pad,   // padding (h, w)
                    l.stride, l.stride,     // stride (h, w)
                    l.dilation, l.dilation, // dilation (h, w)
                    boffset);       // workspace			
		 gemm_ongpu(0, 0, 1, n, k, 1, aoffset, k, boffset, n, 1, coffset, n);
			
		}
	}


	check_error(hipPeekAtLastError());*/
    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, state);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

	int m = l.n;
    activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);


	//cuda_pull_array(l.output_gpu, l.output, l.c*l.out_h*l.out_w);//add by hjimce for debug

	

}


void backward_depthwise_convolutional_layer_gpu(depthwise_convolutional_layer l, network_state state)
{

    constrain_ongpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);


    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, state);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    float *original_input = state.input;

	//cuda_pull_array(net.delta_gpu, net.delta, l.batch*l.c*l.h*l.w);
    /*int m = l.n;
    int n = l.size*l.size;
    int k = l.out_w*l.out_h;
	//pull_depthwise_convolutional_layer(l);//add by hjimce for debug

	for (int b = 0; b < l.batch; ++b) {
		for (int c = 0; c<l.c; c++)
		{



			float *aoffset = l.delta_gpu + c*l.out_h*l.out_w + b*l.n*l.out_h*l.out_w;
			float *boffset = net.workspace;
			float *coffset = l.weight_updates_gpu + c*l.size*l.size;


			float *im = net.input_gpu + c*l.h*l.w + b*l.c*l.h*l.w;


			im2col_gpu(im, 1, l.h, l.w,
				l.size, l.stride, l.pad, boffset);
			gemm_gpu(0, 1, 1, n, k, 1, aoffset, k, boffset, k, 1, coffset, n);


			if (net.delta_gpu) {
				aoffset = l.weights_gpu + c*l.size*l.size;
				boffset = l.delta_gpu + c*l.out_h*l.out_w + b*l.n*l.out_h*l.out_w;
				coffset = net.workspace;

				gemm_gpu(1, 0, n, k, 1, 1, aoffset, n, boffset, k, 0, coffset, k);

				col2im_gpu(net.workspace, 1, l.h, l.w, l.size, l.stride, l.pad, net.delta_gpu + c*l.h*l.w + b*l.n*l.h*l.w);
			}


		}
	}*/
	
	int out_size= l.out_h*l.out_w*l.batch*l.n;
	DepthwiseConv2dBackpropFilterGPUKernelNCHW << <cuda_gridsize(out_size), BLOCK >> > (
		l.delta_gpu, l.stride, l.pad, l.pad, l.out_h, l.out_w, l.c,
		state.input, l.h, l.w, l.n,
		l.weight_updates_gpu, l.size, l.size,
		out_size);
	if (state.delta)
	{
		int in_size = l.h*l.w*l.batch*l.n;
		DepthwiseConv2dBackpropInputGPUKernelNCHW << <cuda_gridsize(in_size), BLOCK >> > (
			l.delta_gpu, l.out_h, l.out_w, l.c,
			l.weights_gpu, l.size, l.size,
			state.delta, l.h, l.w, l.c,
			l.stride, l.pad, l.pad, in_size);

	}
	//cuda_pull_array(net.delta_gpu, net.delta, l.batch*l.c*l.h*l.w);
	//pull_depthwise_convolutional_layer(l);//add by hjimce for debug

//#endif
}
void pull_depthwise_convolutional_layer(depthwise_convolutional_layer layer)
{
   /* cuda_pull_array(layer.weights_gpu, layer.weights, layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.weight_updates_gpu, layer.weight_updates, layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_pull_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_pull_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_pull_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }*/
}

void push_depthwise_convolutional_layer(depthwise_convolutional_layer layer)
{
    /*cuda_push_array(layer.weights_gpu, layer.weights, layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.weight_updates_gpu, layer.weight_updates, layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_push_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_push_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_push_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }*/
}

void update_depthwise_convolutional_layer_gpu(layer l, update_args a)
{
    float learning_rate = a.learning_rate*l.learning_rate_scale;
    float momentum = a.momentum;
    float decay = a.decay;
    int batch = a.batch;

    int size = l.size*l.size*l.c;

    if(a.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, size, batch, a.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        }

    }else{
        axpy_ongpu(size, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_ongpu(size, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_ongpu(size, momentum, l.weight_updates_gpu, 1);

        axpy_ongpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_ongpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_ongpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_ongpu(l.n, momentum, l.scale_updates_gpu, 1);
        }

    }

}


