#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cassert>
#include <vector>
#include <ctime>
#include <chrono>
#include <math.h>
#include <unistd.h>

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <cmath>
#include <fstream>
#include <limits>
#include "rrt_memcpy.hpp"
#include "rfunc.h"
#include <rabbitmq-c/amqp.h>
#include <rabbitmq-c/tcp_socket.h>
#include <string.h>
//#include "utils.h"

//utils.c
#include <ctype.h>
#include <stdarg.h>
#include <rabbitmq-c/framing.h>

#include <assert.h>

#define BLOCK 1024

static hipStream_t streamsArray[4];
static int streamInit[4] = { 0 };

//	ofstream myfile1;
//	  myfile1.open ("rrtGPU_time.txt");

hipStream_t get_cuda_stream_32(int i) {
    if (!streamInit[i]) {
	hipError_t status = hipStreamCreateWithFlags(&streamsArray[i],hipStreamNonBlocking);
        streamInit[i] = 1;
    }
    return streamsArray[i];
}

void Destroy_stream()
{
    for(int i =0;i<4;i++)
    {
	if (streamInit[i]) {
    		hipStreamDestroy(streamsArray[i]);
   		streamInit[i] = 0;
	}
    }
}

double get_time_point() {
    std::chrono::steady_clock::time_point current_time = std::chrono::steady_clock::now();
    return std::chrono::duration_cast<std::chrono::microseconds>(current_time.time_since_epoch()).count();
}


int get_number_of_blocks(int array_size, int block_size)
{
    return array_size / block_size + ((array_size % block_size > 0) ? 1 : 0);
}

__global__ void distance_cal_cuda(vertex* vertex_cuda_ptr,float* new_vertex_cuda,float* distances,int offset)
{
	int idx = blockIdx.x * blockDim.x + offset +  threadIdx.x;
	float current_vertex[2] = {vertex_cuda_ptr[idx].x,vertex_cuda_ptr[idx].y};

	float dx = current_vertex[0] - new_vertex_cuda[0];
	float dy = current_vertex[1] - new_vertex_cuda[1];
	
	distances[idx] = sqrt((dx*dx) + (dy * dy));

}

__device__ bool overlapping_check_dev(float p0, float p3, float p1, float p2)
{
	if (p0 > p3)
	{
	   p0 = p0+p3;
	   p3 = p0 - p3;
	   p0 = p0 - p3;
	}
	if (p1 > p2)
	{
	   p1 = p1+p2;
	   p2 = p1 - p2;
	   p1 = p1 - p2;
	}
	float max_p00_p10 = p0 > p1 ?p0 : p1;
	float min_p30_p20 = p3 < p2 ?p3 : p2;
	return max_p00_p10 <= min_p30_p20;
	
}

__global__ void polygon_collision_check_dev(Polygon* polygons_ ,float* p0,int offset)
{
	int i = blockIdx.x * blockDim.x + offset + threadIdx.x;
	
	__shared__ int flag;
	if(threadIdx.x == 0)
		flag =0;
	__syncthreads();
	
	if(flag == 1 || p0[4] == 1)
	{
		return;
	}
	float p1[2];
  	float p2[2];

	p1[0] = polygons_[i].x1;
	p1[1] = polygons_[i].y1;
  	p2[0] = polygons_[i].x1;
	p2[1] = polygons_[i].y2;

	float cross_p0_p1_p2 = (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
	float cross_p3_p1_p2 = (((p1[0] - p0[2]) * (p2[1] - p0[3])) - ((p2[0] - p0[2]) * (p1[1] - p0[3])));
	float cross_p1_p0_p3 = (((p0[0] - p1[0]) * (p0[3] - p1[1])) - ((p0[2] - p1[0]) * (p0[1] - p1[1])));
	float cross_p2_p0_p3 = (((p0[0] - p2[0]) * (p0[3] - p2[1])) - ((p0[2] - p2[0]) * (p0[1] - p2[1])));
	int sign_cross_p0_p1_p2 =  cross_p0_p1_p2 >=0 ? cross_p0_p1_p2 ? 1 :0 :-1;
	int sign_cross_p3_p1_p2 =  cross_p3_p1_p2 >=0 ? cross_p3_p1_p2 ? 1 :0 :-1;
	int sign_cross_p1_p0_p3 =  cross_p1_p0_p3 >=0 ? cross_p1_p0_p3 ? 1 :0 :-1;
	int sign_cross_p2_p0_p3 =  cross_p2_p0_p3 >=0 ? cross_p2_p0_p3 ? 1 :0 :-1;	

	bool overlapping_check_x = overlapping_check_dev(p0[0],p0[2],p1[0],p2[0]);
	bool overlapping_check_y = overlapping_check_dev(p0[1],p0[3],p1[1],p2[1]);
	
	if((cross_p0_p1_p2 == 0) &&  (cross_p3_p1_p2 == 0)) // Lines are colinear
	{
		if (overlapping_check_x || overlapping_check_y) //lines are overlapping
		{
			flag = 1;
			p0[4] = 1;
			return;	
		}
			
	}
	
	if((sign_cross_p0_p1_p2 != sign_cross_p3_p1_p2) && (sign_cross_p1_p0_p3 != sign_cross_p2_p0_p3))    //lines are intersecting
	{
		flag = 1;
		p0[4] = 1;
		return;	
	}
	if(flag == 1 || p0[4] == 1)
	{
		return;
	}
	p1[0] = polygons_[i].x1;
	p1[1] = polygons_[i].y2;
  	p2[0] = polygons_[i].x2;
	p2[1] = polygons_[i].y2;

	cross_p0_p1_p2 = (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
	cross_p3_p1_p2 = (((p1[0] - p0[2]) * (p2[1] - p0[3])) - ((p2[0] - p0[2]) * (p1[1] - p0[3])));
	cross_p1_p0_p3 = (((p0[0] - p1[0]) * (p0[3] - p1[1])) - ((p0[2] - p1[0]) * (p0[1] - p1[1])));
	cross_p2_p0_p3 = (((p0[0] - p2[0]) * (p0[3] - p2[1])) - ((p0[2] - p2[0]) * (p0[1] - p2[1])));
	sign_cross_p0_p1_p2 =  cross_p0_p1_p2 >=0 ? cross_p0_p1_p2 ? 1 :0 :-1;
	sign_cross_p3_p1_p2 =  cross_p3_p1_p2 >=0 ? cross_p3_p1_p2 ? 1 :0 :-1;
	sign_cross_p1_p0_p3 =  cross_p1_p0_p3 >=0 ? cross_p1_p0_p3 ? 1 :0 :-1;
	sign_cross_p2_p0_p3 =  cross_p2_p0_p3 >=0 ? cross_p2_p0_p3 ? 1 :0 :-1;	

	overlapping_check_x = overlapping_check_dev(p0[0],p0[2],p1[0],p2[0]);
	overlapping_check_y = overlapping_check_dev(p0[1],p0[3],p1[1],p2[1]);
		
	if((cross_p0_p1_p2 == 0) &&  (cross_p3_p1_p2 == 0)) // Lines are colinear
	{
		if (overlapping_check_x || overlapping_check_y) //lines are overlapping
		{
			flag = 1;
			p0[4] = 1;
			return;	
		}
			
	}
	
	if((sign_cross_p0_p1_p2 != sign_cross_p3_p1_p2) && (sign_cross_p1_p0_p3 != sign_cross_p2_p0_p3))    //lines are intersecting
	{
		flag = 1;
		p0[4] = 1;
		return;	
	}
	
	if(flag == 1 || p0[4] == 1)
	{
		return;
	}
	p1[0] = polygons_[i].x2;
	p1[1] = polygons_[i].y2;
  	p2[0] = polygons_[i].x2;
	p2[1] = polygons_[i].y1;
	
	cross_p0_p1_p2 = (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
	cross_p3_p1_p2 = (((p1[0] - p0[2]) * (p2[1] - p0[3])) - ((p2[0] - p0[2]) * (p1[1] - p0[3])));
	cross_p1_p0_p3 = (((p0[0] - p1[0]) * (p0[3] - p1[1])) - ((p0[2] - p1[0]) * (p0[1] - p1[1])));
	cross_p2_p0_p3 = (((p0[0] - p2[0]) * (p0[3] - p2[1])) - ((p0[2] - p2[0]) * (p0[1] - p2[1])));
	sign_cross_p0_p1_p2 =  cross_p0_p1_p2 >=0 ? cross_p0_p1_p2 ? 1 :0 :-1;
	sign_cross_p3_p1_p2 =  cross_p3_p1_p2 >=0 ? cross_p3_p1_p2 ? 1 :0 :-1;
	sign_cross_p1_p0_p3 =  cross_p1_p0_p3 >=0 ? cross_p1_p0_p3 ? 1 :0 :-1;
	sign_cross_p2_p0_p3 =  cross_p2_p0_p3 >=0 ? cross_p2_p0_p3 ? 1 :0 :-1;	

	overlapping_check_x = overlapping_check_dev(p0[0],p0[2],p1[0],p2[0]);
	overlapping_check_y = overlapping_check_dev(p0[1],p0[3],p1[1],p2[1]);
		
	if((cross_p0_p1_p2 == 0) &&  (cross_p3_p1_p2 == 0)) // Lines are colinear
	{
		if (overlapping_check_x || overlapping_check_y) //lines are overlapping
		{
			flag = 1;
			p0[4] = 1;
			return;	
		}
			
	}
	
	if((sign_cross_p0_p1_p2 != sign_cross_p3_p1_p2) && (sign_cross_p1_p0_p3 != sign_cross_p2_p0_p3))    //lines are intersecting
	{
		flag = 1;
		p0[4] = 1;
		return;	
	}	
	if(flag == 1 || p0[4] == 1)
	{
		return;
	}
	p1[0] = polygons_[i].x2;
	p1[1] = polygons_[i].y1;
  	p2[0] = polygons_[i].x1;
	p2[1] = polygons_[i].y1;
	
	cross_p0_p1_p2 = (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
	cross_p3_p1_p2 = (((p1[0] - p0[2]) * (p2[1] - p0[3])) - ((p2[0] - p0[2]) * (p1[1] - p0[3])));
	cross_p1_p0_p3 = (((p0[0] - p1[0]) * (p0[3] - p1[1])) - ((p0[2] - p1[0]) * (p0[1] - p1[1])));
	cross_p2_p0_p3 = (((p0[0] - p2[0]) * (p0[3] - p2[1])) - ((p0[2] - p2[0]) * (p0[1] - p2[1])));
	sign_cross_p0_p1_p2 =  cross_p0_p1_p2 >=0 ? cross_p0_p1_p2 ? 1 :0 :-1;
	sign_cross_p3_p1_p2 =  cross_p3_p1_p2 >=0 ? cross_p3_p1_p2 ? 1 :0 :-1;
	sign_cross_p1_p0_p3 =  cross_p1_p0_p3 >=0 ? cross_p1_p0_p3 ? 1 :0 :-1;
	sign_cross_p2_p0_p3 =  cross_p2_p0_p3 >=0 ? cross_p2_p0_p3 ? 1 :0 :-1;	
	overlapping_check_x = overlapping_check_dev(p0[0],p0[2],p1[0],p2[0]);
	overlapping_check_y = overlapping_check_dev(p0[1],p0[3],p1[1],p2[1]);
		
	if((cross_p0_p1_p2 == 0) &&  (cross_p3_p1_p2 == 0)) // Lines are colinear
	{
		if (overlapping_check_x || overlapping_check_y) //lines are overlapping
		{
			flag = 1;
			p0[4] = 1;
			return;	
		}
			
	}
	
	if((sign_cross_p0_p1_p2 != sign_cross_p3_p1_p2) && (sign_cross_p1_p0_p3 != sign_cross_p2_p0_p3))    //lines are intersecting
	{
		flag = 1;
		p0[4] = 1;
		return;	
	}
	
}

float distance_(const float *p1, const float *p2)
{
  const float dx = p1[0] - p2[0];
  const float dy = p1[1] - p2[1];
  

  return sqrt(pow(dx, 2) + pow(dy, 2));
}

int min_distance(float* distance_list,int vertex_count)
{
	float min_dstance = distance_list[0];
	int idx = 0;	
	for(int i=1; i <vertex_count;i++)
	{
		if(distance_list[i] < min_dstance)
		{
			min_dstance = distance_list[i];
			idx = i;
		} 
	}

	return idx;
}


float cross_(const float *p0, const float *p1, const float *p2)
{
	return (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
}

int sign_(float value)
{
	return value >=0 ? value ? 1 :0 :-1;
}

float max_(float p0 , float p1)
{
	return p0 > p1?p0 : p1;
}

float min_(float p0 , float p1)
{
	return p0 < p1?p0 : p1;
}


bool overlapping_check(float p0, float p3, float p1, float p2)
{
	if (p0 > p3)
	{
	   p0 = p0+p3;
	   p3 = p0 - p3;
	   p0 = p0 - p3;
	}
	if (p1 > p2)
	{
	   p1 = p1+p2;
	   p2 = p1 - p2;
	   p1 = p1 - p2;
	}
	return max_(p0,p1) <= min_(p3,p2);
	
}

RRT::RRT(float *start, float *goal, int rando)
        : start_(start),
          goal_(goal),
          delta_(0.1),
          epsilon_(0),
          xmin_(0),
          xmax_(1000),
          ymin_(0),
          ymax_(1000),
          resolution_(1.0),
          max_iter_(MAX_ITER),
          vertex_count_(0),
	  obstacles_count_(0)
{
  std::srand(rando);
}

bool RRT::polygon_collision_check_cuda(const vertex &v_new,const vertex &v_near,int goal_check)
{
//	#ifdef WR2F
//	high_resolution_clock::time_point t1 = high_resolution_clock::now();
//	#endif	
	static double total_time1 = 0;
  	static int count1 = 0;
	double begin_time = get_time_point();
	v_new_near_host[0] = v_new.x;
	v_new_near_host[1] = v_new.y;	
	v_new_near_host[2] = v_near.x;
	v_new_near_host[3] = v_near.y;
	v_new_near_host[4] = 0;	
	
	////////////////////////////////////////////////////////////////
	int num_blocks = get_number_of_blocks(obstacles_count_, BLOCK);
	int extra_blocks = num_blocks %4;
	int stream_id = 0;
	num_blocks = num_blocks /4;
	int loopcount = 0;
	int offset = 0;
	for (loopcount = 0; loopcount < num_blocks;loopcount++)
	{
		offset = loopcount * 4*BLOCK;
		stream_id = loopcount%4;
		polygon_collision_check_dev<<<4, BLOCK, 0, get_cuda_stream_32(stream_id)>>>(polygons_dev,v_new_near_dev,offset);	
	}
	if(extra_blocks)
	{
		stream_id = loopcount%4;
		offset = loopcount * 4*BLOCK;
		polygon_collision_check_dev<<<extra_blocks, BLOCK, 0, get_cuda_stream_32(stream_id)>>>(polygons_dev,v_new_near_dev,offset);	
	}
	if(loopcount == 0)
	{
		hipEventRecord(stopEvent,get_cuda_stream_32(0));
		while(hipEventQuery(stopEvent) != 0);
	}
	else
	{
		hipDeviceSynchronize();
	}
	////////////////////////////////////////////////////////////////

	/*int blocksize_ = obstacles_count_ > 1024?1024:obstacles_count_ ;
	int gridsize_ = obstacles_count_ % 1024?int(obstacles_count_/1024) + 1:int(obstacles_count_/1024);
        int offset = 0;	*/
//	#ifdef WR2F
//	high_resolution_clock::time_point t2 = high_resolution_clock::now();
//	duration<double, milli> time_span = t2-t1;
//	myfile1 <<"Obstacle Detection = "  << time_span.count() << endl; 
//	#endif	
	count1++;
	double time_diff = ((double)get_time_point() - begin_time) / 1000;
	total_time1+= time_diff;
	if(v_new_near_host[4])
	{	
		return true;
	}
	else
	{
		if(goal_check)
			printf("Average Time for collision check is %lf milli-seconds \n", total_time1/(count1));
		return false;
	}
}


bool RRT::PolygonCollisionCheck(const vertex &v_new, const vertex &v_near)
{

  
  const float p0[2] = {v_new.x, v_new.y};
  const float p3[2] = {v_near.x, v_near.y};
  
  float p1[2];
  float p2[2];

  for(unsigned int i = 0; i < obstacles_count_; i++)
  {
	p1[0] = polygons_[i].x1;
	p1[1] = polygons_[i].y1;
  	p2[0] = polygons_[i].x1;
	p2[1] = polygons_[i].y2;
	
	if((cross_(p0, p1, p2) == 0) &&  (cross_(p3, p1, p2) == 0)) // Lines are colinear
	{
		if (overlapping_check(p0[0], p3[0], p1[0], p2[0]) || overlapping_check(p0[1], p3[1], p1[1], p2[1])) //lines are overlapping
		{
			return true; 	
		}
			
	}
	
	if((sign_(cross_(p0, p1, p2)) != sign_(cross_(p3, p1, p2))) && (sign_(cross_(p1, p0, p3)) != sign_(cross_(p2, p0, p3))))    //lines are intersecting
	{

			return true;
	}
	
	/*if ((i==1249))
	{
		printf(" CPU %d - %f ,%f , %f, %f \n",i,cross_(p0, p1, p2),cross_(p3, p1, p2),cross_(p1, p0, p3),cross_(p2, p0, p3));
		printf("CPU - %d, p1 = %f %f, p2 = %f %f, p0 = %f %f, p3 = %f %f \n ",i,p1[0],p1[1],p2[0],p2[1],p0[0],p0[1],p3[0],p3[1]);
	}*/
		
	
	p1[0] = polygons_[i].x1;
	p1[1] = polygons_[i].y2;
  	p2[0] = polygons_[i].x2;
	p2[1] = polygons_[i].y2;
	
	if((cross_(p0, p1, p2) == 0) &&  (cross_(p3, p1, p2) == 0)) // Lines are colinear
	{
		if (overlapping_check(p0[0], p3[0], p1[0], p2[0]) || overlapping_check(p0[1], p3[1], p1[1], p2[1])) //lines are overlapping
		{
			return true; 	
		}
			
	}
	
	if((sign_(cross_(p0, p1, p2)) != sign_(cross_(p3, p1, p2))) && (sign_(cross_(p1, p0, p3)) != sign_(cross_(p2, p0, p3))))    //lines are intersecting
	{
			return true;
		
	}	
	
	p1[0] = polygons_[i].x2;
	p1[1] = polygons_[i].y2;
  	p2[0] = polygons_[i].x2;
	p2[1] = polygons_[i].y1;
	
	if((cross_(p0, p1, p2) == 0) &&  (cross_(p3, p1, p2) == 0)) // Lines are colinear
	{
		if (overlapping_check(p0[0], p3[0], p1[0], p2[0]) || overlapping_check(p0[1], p3[1], p1[1], p2[1])) //lines are overlapping
		{
			return true; 	
		}
			
	}
	
	if((sign_(cross_(p0, p1, p2)) != sign_(cross_(p3, p1, p2))) && (sign_(cross_(p1, p0, p3)) != sign_(cross_(p2, p0, p3))))    //lines are intersecting
	{
			return true;
		
	}

	p1[0] = polygons_[i].x2;
	p1[1] = polygons_[i].y1;
  	p2[0] = polygons_[i].x1;
	p2[1] = polygons_[i].y1;
	
	if((cross_(p0, p1, p2) == 0) &&  (cross_(p3, p1, p2) == 0)) // Lines are colinear
	{
		if (overlapping_check(p0[0], p3[0], p1[0], p2[0]) || overlapping_check(p0[1], p3[1], p1[1], p2[1])) //lines are overlapping
		{
			return true;	
		}
			
	}
	
	if((sign_(cross_(p0, p1, p2)) != sign_(cross_(p3, p1, p2))) && (sign_(cross_(p1, p0, p3)) != sign_(cross_(p2, p0, p3))))    //lines are intersecting
	{
			return true;
		
	}
  }
  return false;
}

bool RRT::exploreObstacles()
{

  vertex_count_ = 0;
  //vertices_.clear();
  // add start to graph
  vertex v_start;
  v_start.x = start_[0];
  v_start.y = start_[1];
  addVertex(v_start);


  bool success = false;
  int ctr = 0;


  while(!success)
  {
	
    if (ctr > max_iter_)
    {
      std::cout << "Goal not achieved" << std::endl;
      return false;
    }

    // std::cout << "Iter: " << ctr << std::endl;
	float rand[2];
    // 1) random point
    randomConfig(rand);
// std::cout << "randat: " << rand_point_cuda[0] << ", " << rand_point_cuda[1] <<std::endl;

    // 2) nearest node in graph
    vertex v_near;
    high_resolution_clock::time_point t3 = high_resolution_clock::now();
     nearestVertex(v_near, rand);
    high_resolution_clock::time_point t4 = high_resolution_clock::now();
    duration<double, milli> time_span = t4-t3;
    #ifdef WR2F
    myfile <<"Nearest Vertex = "  << time_span.count() << endl; 
    #endif 
    // 3) new node
    vertex v_new;
    if(!newConfiguration(v_new, v_near, rand))
    {
      continue;
    }

//    std::cout << "v new at: " << v_new.x << ", " << v_new.y <<std::endl;

    ctr++;

    // 4) check for collisions
   // if (collision_check(v_new, v_near))
   // {
      // std::cout << "Collision" << std::endl;
   //   continue;
   // }
   t3 = high_resolution_clock::now();
   bool collision_flag = PolygonCollisionCheck(v_new, v_near);
   t4 = high_resolution_clock::now();
   time_span = t4-t3;
    #ifdef WR2F
    myfile <<"Obstacle Detection = "  << time_span.count() << endl;  
   #endif
   //cout <<  "CPU Time " << time_span.count() << " ms" << endl; 
    if (collision_flag)
    {
	//std::cout << "Collision" << std::endl;
       continue;
    }

    // std::cout << v_new.x << " " << v_new.y << "\n";

    // 6) add new node
    addVertex(v_new);
    addEdge(v_near, v_new);

    // 7) win check
  t3 = high_resolution_clock::now();
  bool win_flag = win_check(v_new, goal_);
  t4 = high_resolution_clock::now();
   time_span = t4-t3;
   #ifdef WR2F
   myfile <<"Obstacle Detection = "  << time_span.count() << endl; 
   #endif
  // cout <<  "CPU Time " << time_span.count() << " ms" << endl; 
  //  win_flag = 0;
    if (win_flag)
    {
      std::cout << "Goal reached on CPU" << std::endl;
      printf("ctr = %d \n",ctr);
      // add goal to graph
      vertex v_goal;
      v_goal.x = goal_[0];
      v_goal.y = goal_[1];
      addVertex(v_goal);
      addEdge(v_new, v_goal);
      std::cout << "Vnew " << v_new.x << " " << v_new.y << "\n";
      std::cout << "Goal " << v_goal.x << " " << v_goal.y << "\n";
      success = true;
      break;
    }

  }

  return success;
}


bool RRT::win_check(const vertex &v_new, const float *goal)
{
  //cast goal to vertex //TODO: overlead collision to optionally take float as second arg
  vertex v_goal(goal[0],goal[1]);
  // std::cout << "SURUR\n";
  bool collis_check = PolygonCollisionCheck(v_new, v_goal);

 // std::cout << collis_check << std::endl;

  return !collis_check;
}

bool RRT::win_check_cuda(const vertex &v_new,const float *goal)
{
  vertex v_goal(goal[0],goal[1]);
  bool collis_check = polygon_collision_check_cuda(v_new,v_goal,1);
//PolygonCollisionCheck(v_new, v_goal,polygons_dev);

 // std::cout << collis_check << std::endl;

  return !collis_check;
}

bool RRT::exploreCuda()
{
  static double total_time = 0;
  static int count = 0;
  double time_diff;
   vertex_count_ = 0;
 // vertices_.clear();
   // add start to graph
  vertex v_start;
  v_start.x = start_[0];
  v_start.y = start_[1];
  addVertex(v_start);
  bool success = false;
  int ctr = 0;
  while(!success)
  {
	
    if (ctr > max_iter_)
    {
      std::cout << "Goal not achieved" << std::endl;
      return false;
    }
    // printf("ctr = %d\n",ctr);
    float rand[2];
    randomConfig(rand);
    vertex v_near;
 //   high_resolution_clock::time_point t3 = high_resolution_clock::now();
    double begin_time = get_time_point();
    if(vertex_count_ > 1024)
    	nearestVertex_cuda(v_near, rand);
    else
     	nearestVertex(v_near, rand);
    count++;
    time_diff = ((double)get_time_point() - begin_time) / 1000;
    total_time+= time_diff;
   // high_resolution_clock::time_point t4 = high_resolution_clock::now();
  //  duration<double, milli> time_span = t4-t3;
//    #ifdef WR2F
//    myfile1 <<"Nearest Vertex = "  << time_span.count() << endl; 
//    #endif
    vertex v_new;
	
    if(!newConfiguration(v_new, v_near, rand))
    {
	
      continue;
    }
    ctr++;
	
    if (polygon_collision_check_cuda(v_new,v_near,0))
    {
       continue;
    }
    addVertex(v_new);
    addEdge(v_near, v_new);

    bool win_flag = win_check_cuda(v_new,goal_);
    if (win_flag)
    {
	printf("Average Time for Nearest Vertex find is %lf milli-seconds \n", total_time/(count));
     // std::cout << "Goal reached on GPU" << std::endl;
  //    printf("ctr = %d \n",ctr);
      vertex v_goal;
      v_goal.x = goal_[0];
      v_goal.y = goal_[1];
      addVertex(v_goal);
      addEdge(v_new, v_goal);
     // std::cout << "Vnew " << v_new.x << " " << v_new.y << "\n";
    //  std::cout << "Goal " << v_goal.x << " " << v_goal.y << "\n";
      success = true;
      break;
    }

  }

  return success;
}


void RRT::randomObstacles(int num_obstacle, float x_min, float x_max, float y_min, float y_max)
{
	for(int i = 0; i<num_obstacle;i++)
	{
		 float x1 = xmin_+static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(xmax_-xmin_)));
    		 float y1 = ymin_+static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(ymax_-ymin_)));
		 float x2 = x1 + x_min +static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(x_max-x_min)));
    		 float y2 = y1 +y_min + +static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(y_max- y_min)));

		if ((start_[0] >= x1) && (start_[0] <= x2) && (start_[1] >= y1) && (start_[1] <= y2))
		{
			i--;
			//printf("%f %f %f %f start within the polygon\n ",x1,y1,x2,y2);
			continue;
		}
		if (goal_[0] >= x1 && goal_[0] <= x2 && goal_[1] >= y1 && goal_[1] <= y2)
		{
			i--;
			//printf("%f %f %f %f goal within the polygon\n ",x1,y1,x2,y2);	
			continue;
		}
		Polygon poly;
		poly.x1 = x1;
		poly.y1 = y1;
		poly.x2 = x2;	
		poly.y2 = y2;
		//polygons_.push_back(poly);
		//polygons_dev[i] = poly;
		polygons_[obstacles_count_++] = poly;
		//printf(" polygo= %f %f %f %f \n",x1,y1,x2,y2);
		
	}
	//printf("size = %d\n",polygons_.size());
}

void RRT::addVertex(vertex &v)
{
  v.id = vertex_count_;

  vertex_host_ptr[vertex_count_++] = v;
  

  // std::cout << "New vertex count: " << vertex_count_ << std::endl;
} 



void RRT::addEdge(const vertex &v_near, const vertex &v_new)
{
  // search for node1 and node2
  // addes edge btw both
  bool added = false;


  for(unsigned int i = 0; i < vertex_count_; i++)
  {
    // found node 1
    if (vertex_host_ptr[i].id == v_near.id)
    {
      for(unsigned int j = 0; j < vertex_count_; j++)
      {
        // do not add vertex to itself
        // found node 2
        if(vertex_host_ptr[j].id == v_new.id && i != j)
        {
          // edge connecting node 1 to node 2
          // std::cout << "adding edge " << v_near.id << "->" << v_new.id << std::endl;
          // v_near.adjacent_vertices.push_back(v_new.id);
          vertex_host_ptr[v_near.id].adjacent_vertices.push_back(v_new.id);
          added = true;
        }

      } // end inner loop
    }
  } // end outer loop

  if (!added)
  {
    std::cout << "Error: 'addEdge' edge not added" << std::endl;
  }
}


bool RRT::newConfiguration(vertex &v_new, const vertex &v_near, const float *q_rand) const
{

  //std::cout << "Vnear " << v_near.x << " " << v_near.y << "\n";
  // difference btw q_rand and v_near
  const float vx = q_rand[0] - v_near.x;
  const float vy = q_rand[1] - v_near.y;
  
//std::cout << "Vx " << vx << " " << vy << "\n";
  // distance_ between v_near and q_rand
  const float magnitude = std::sqrt(std::pow(vx, 2) + std::pow(vy, 2));
  
//  std::cout << "Magnitude " << magnitude << std::endl;

  if (magnitude == 0)
  {
    return false;
  }

  // unit vector in driection of q_rand
  const float ux = vx / magnitude;
  const float uy = vy / magnitude;

  // place v_new a delta away from v_near
  v_new.x = v_near.x + delta_ * ux;
  v_new.y = v_near.y + delta_ * uy;
 //std::cout << "ux " << ux << " " << uy << "\n";
 // std::cout << "Vnew " << v_new.x << " " << v_new.y << "\n";

  // make sure still within bounds
  if (v_new.x > xmax_ || v_new.x < xmin_ || v_new.y > ymax_ || v_new.y < ymin_)
  {
    return false;
  }

  return true;
}



void RRT::nearestVertex(vertex &v, float *q_rand) const
{
  float point[2];
  std::vector<float> d;

  for(unsigned int i = 0; i < vertex_count_; i++)
  {
    point[0] = vertex_host_ptr[i].x;
    point[1] = vertex_host_ptr[i].y;

    distances_host[i] = (distance_(point, q_rand));
  }

  int idx = min_distance(distances_host,vertex_count_);
  v = vertex_host_ptr[idx];
}

void RRT::nearestVertex_cuda(vertex &v, float *q_rand) const
{
	
	/*int blocksize_ = vertex_count_ > 1024?1024:vertex_count_ ;
	int gridsize_ = vertex_count_ % 1024?int(vertex_count_/1024) + 1:int(vertex_count_/1024);
	dim3 blocksize(blocksize_);
	dim3 gridsize(gridsize_);*/
	new_vertex_host[0] = q_rand[0];
	new_vertex_host[1] = q_rand[1];
	//////////////////////////////////////////////////////////////////////////////
	int num_blocks = get_number_of_blocks(vertex_count_, BLOCK);
	int extra_blocks = num_blocks %4;
	int stream_id = 0;
	num_blocks = num_blocks /4;
	int loopcount = 0;
	int offset = 0;
	for (loopcount = 0; loopcount < num_blocks;loopcount++)
	{
		offset = loopcount * 4*BLOCK;
		stream_id = loopcount%4;
		distance_cal_cuda<<<4, BLOCK, 0, get_cuda_stream_32(stream_id)>>>(vertex_cuda_ptr,new_vertex_cuda,distances_cuda,offset);	
	}
	if(extra_blocks)
	{
		stream_id = loopcount%4;
		offset = loopcount * 4*BLOCK;
		distance_cal_cuda<<<extra_blocks, BLOCK, 0, get_cuda_stream_32(stream_id)>>>(vertex_cuda_ptr,new_vertex_cuda,distances_cuda,offset);	
	}
	if(loopcount == 0)
	{
		hipEventRecord(stopEvent,get_cuda_stream_32(0));
		while(hipEventQuery(stopEvent) != 0);
	}
	else
	{
		hipDeviceSynchronize();
	}
	////////////////////////////////////////////////////////////////////////////////


	/*distance_cal_cuda<<<gridsize,blocksize,0,get_cuda_stream_32(0)>>>(vertex_cuda_ptr,new_vertex_cuda,distances_cuda,offset);
	hipEventRecord(stopEvent,stream[0]);
	while(hipEventQuery(stopEvent) != 0);*/
	int idx = min_distance(distances_host,vertex_count_);
	v = vertex_host_ptr[idx];
	
}


void RRT::randomConfig(float *q_rand) const
{
  // x position
  q_rand[0] = xmin_+static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(xmax_-xmin_)));

  // y position
  q_rand[1] = ymin_+static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(ymax_-ymin_)));
}

int RRT::findParent(const vertex &v) const
{
  // iterate over vertices
  for(unsigned int i = 0; i < vertex_count_; i++)
  {
    for(unsigned int j = 0; j < vertex_host_ptr[i].adjacent_vertices.size(); j++)
    {
      if (vertex_host_ptr[i].adjacent_vertices.at(j) == v.id)
      {
        // std::cout << "Parent found" << std::endl;
        return i;
      }
    } // end inner loop
  } // end outer loop

  std::cout << "Parent not found" << std::endl;
  return -1;
}

void destroy_cuda_mem(void)
{
 // hipFree(polygons_dev);
  hipFree(v_new_near_dev);
  hipHostFree(polygons_);
  hipHostFree(v_new_near_host);
  hipFree(vertex_cuda_ptr);
  hipFree(new_vertex_cuda);
  hipFree(distances_cuda);
  hipHostFree(vertex_host_ptr);
  hipHostFree(new_vertex_host);
  hipHostFree(distances_host);
  hipEventDestroy(stopEvent);

}

void die(const char *fmt, ...) {
  va_list ap;
  va_start(ap, fmt);
  vfprintf(stderr, fmt, ap);
  va_end(ap);
  fprintf(stderr, "\n");
  exit(1);
}

void die_on_error(int x, char const *context) {
  if (x < 0) {
    fprintf(stderr, "%s: %s\n", context, amqp_error_string2(x));
    exit(1);
  }
}

void die_on_amqp_error(amqp_rpc_reply_t x, char const *context) {
  switch (x.reply_type) {
    case AMQP_RESPONSE_NORMAL:
      return;

    case AMQP_RESPONSE_NONE:
      fprintf(stderr, "%s: missing RPC reply type!\n", context);
      break;

    case AMQP_RESPONSE_LIBRARY_EXCEPTION:
      fprintf(stderr, "%s: %s\n", context, amqp_error_string2(x.library_error));
      break;

    case AMQP_RESPONSE_SERVER_EXCEPTION:
      switch (x.reply.id) {
        case AMQP_CONNECTION_CLOSE_METHOD: {
          amqp_connection_close_t *m =
              (amqp_connection_close_t *)x.reply.decoded;
          fprintf(stderr, "%s: server connection error %uh, message: %.*s\n",
                  context, m->reply_code, (int)m->reply_text.len,
                  (char *)m->reply_text.bytes);
          break;
        }
        case AMQP_CHANNEL_CLOSE_METHOD: {
          amqp_channel_close_t *m = (amqp_channel_close_t *)x.reply.decoded;
          fprintf(stderr, "%s: server channel error %uh, message: %.*s\n",
                  context, m->reply_code, (int)m->reply_text.len,
                  (char *)m->reply_text.bytes);
          break;
        }
        default:
          fprintf(stderr, "%s: unknown server error, method id 0x%08X\n",
                  context, x.reply.id);
          break;
      }
      break;
  }

  exit(1);
}

static void dump_row(long count, int numinrow, int *chs) {
  int i;

  printf("%08lX:", count - numinrow);

  if (numinrow > 0) {
    for (i = 0; i < numinrow; i++) {
      if (i == 8) {
        printf(" :");
      }
      printf(" %02X", chs[i]);
    }
    for (i = numinrow; i < 16; i++) {
      if (i == 8) {
        printf(" :");
      }
      printf("   ");
    }
    printf("  ");
    for (i = 0; i < numinrow; i++) {
      if (isprint(chs[i])) {
        printf("%c", chs[i]);
      } else {
        printf(".");
      }
    }
  }
  printf("\n");
}

static int rows_eq(int *a, int *b) {
  int i;

  for (i = 0; i < 16; i++)
    if (a[i] != b[i]) {
      return 0;
    }

  return 1;
}

void amqp_dump(void const *buffer, size_t len) {
  unsigned char *buf = (unsigned char *)buffer;
  long count = 0;
  int numinrow = 0;
  int chs[16];
  int oldchs[16] = {0};
  int showed_dots = 0;
  size_t i;

  for (i = 0; i < len; i++) {
    int ch = buf[i];

    if (numinrow == 16) {
      int j;

      if (rows_eq(oldchs, chs)) {
        if (!showed_dots) {
          showed_dots = 1;
          printf(
              "          .. .. .. .. .. .. .. .. : .. .. .. .. .. .. .. ..\n");
        }
      } else {
        showed_dots = 0;
        dump_row(count, numinrow, chs);
      }

      for (j = 0; j < 16; j++) {
        oldchs[j] = chs[j];
      }

      numinrow = 0;
    }

    count++;
    chs[numinrow++] = ch;
  }

  dump_row(count, numinrow, chs);

  if (numinrow != 0) {
    printf("%08lX:\n", count);
  }
}


int main(int argc, char * argv[])
{

 ///  cpu_set_t cpuset_sched;
//CPU_ZERO(&cpuset_sched);
//CPU_SET(4,&cpuset_sched);
//sched_setaffinity(getpid(),sizeof(cpuset_sched),&cpuset_sched);

  printf("\n\n**===-------------------------------------------------===**\n");
  printf("\n\n**===--- RRT ---===**\n");
  printf("\n\n**===-------------------------------------------------===**\n");
  float start[] = {0,0};
  float goal[] = {1000,1000};//{float(rand()%1000),float(rand()%1000)};
  
//	FILE *fp;
//	fp = fopen("RRT_time.txt", "w");

  //expecting command line args to be rand num, then num onbstacles
  if (argc > 2)
  {
    num_obstacles = std::atoi(argv[2]);
    rand_num = std::atoi(argv[1]);
  } else if (argc > 1) {
    rand_num = std::atoi(argv[1]);
    num_obstacles = 2048;
  } else {
    num_obstacles = 2048;
    rand_num = 20;
  }

  RRT rrt(start, goal, rand_num);
  hipHostAlloc(&polygons_,num_obstacles*sizeof(Polygon),hipHostMallocMapped);
  rrt.randomObstacles(num_obstacles, 1.0, 2.0, 1.0, 2.0);
  hipHostAlloc(&v_new_near_host,5*sizeof(float),hipHostMallocMapped);
  status = hipHostGetDevicePointer(&v_new_near_dev,v_new_near_host,0);  
  hipHostAlloc(&vertex_host_ptr,MAX_ITER*sizeof(vertex),hipHostMallocMapped);
  hipHostAlloc(&new_vertex_host,2*sizeof(float),hipHostMallocMapped);
  hipHostAlloc(&distances_host,MAX_ITER*sizeof(float),hipHostMallocMapped);
  status = hipHostGetDevicePointer(&distances_cuda,distances_host,0);  
  status = hipHostGetDevicePointer(&new_vertex_cuda,new_vertex_host,0);  
  status = hipHostGetDevicePointer(&vertex_cuda_ptr,vertex_host_ptr,0);  
  hipEventCreateWithFlags(&stopEvent,hipEventDisableTiming);
  status = hipHostGetDevicePointer(&polygons_dev,polygons_,0);  
  double total_time = 0; 
  int loopcount = 0;
//  int path_deadline_missed = 0;
//  char *s1;

  rfunc_init ();

//	ofstream myfile1;
//	  myfile1.open ("rrtGPU_time.txt");

//	  char const *hostname;
//	  int port, status;
//	  char const *exchange;
//	  char const *bindingkey;
//	  amqp_socket_t *socket = NULL;
//	  amqp_connection_state_t conn;

//	  amqp_bytes_t queuename;

//	  hostname = "localhost";
//	  port = 5672;
//	  exchange = "amq.direct";   /* argv[3]; */
//	  bindingkey = "test"; /* argv[4]; */

//  conn = amqp_new_connection();

//  socket = amqp_tcp_socket_new(conn);
//  if (!socket) {
//    die("creating TCP socket");
//  }

//  status = amqp_socket_open(socket, hostname, port);
//  if (status) {
//    die("opening TCP socket");
//  }

//  die_on_amqp_error(amqp_login(conn, "/", 0, 131072, 0, AMQP_SASL_METHOD_PLAIN,
//                               "guest", "guest"),
//                    "Logging in");
//  amqp_channel_open(conn, 1);
//  die_on_amqp_error(amqp_get_rpc_reply(conn), "Opening channel");

//  {
//    amqp_queue_declare_ok_t *r = amqp_queue_declare(
//        conn, 1, amqp_empty_bytes, 0, 0, 0, 1, amqp_empty_table);
//    die_on_amqp_error(amqp_get_rpc_reply(conn), "Declaring queue");
//	    queuename = amqp_cstring_bytes("slamst");
//  }

//  amqp_queue_bind(conn, 1, queuename, amqp_cstring_bytes(exchange),
//                  amqp_cstring_bytes(bindingkey), amqp_empty_table);
//  die_on_amqp_error(amqp_get_rpc_reply(conn), "Binding queue");

//  amqp_basic_consume(conn, 1, queuename, amqp_empty_bytes, 0, 1, 0,
//                     amqp_empty_table);
//  die_on_amqp_error(amqp_get_rpc_reply(conn), "Consuming");

//	    queuename = amqp_cstring_bytes("slamst");

//	double begin_time=0;

//forconsumer.c code

//  amqp_frame_t frame;
//    amqp_rpc_reply_t ret;
//    amqp_envelope_t envelope;


  for(loopcount = 0; loopcount < 200;loopcount++)
  {

	  rfunc();	

/////////listen.c code
//	amqp_rpc_reply_t res;
//      amqp_envelope_t envelope;

//      amqp_maybe_release_buffers(conn);

//      res = amqp_consume_message(conn, &envelope, NULL, 0);

//      if (AMQP_RESPONSE_NORMAL != res.reply_type) {
//        break;
//      }

//      printf("Delivery %u, exchange %.*s routingkey %.*s\n",
//             (unsigned)envelope.delivery_tag, (int)envelope.exchange.len,
//             (char *)envelope.exchange.bytes, (int)envelope.routing_key.len,
//             (char *)envelope.routing_key.bytes);

//      if (envelope.message.properties._flags & AMQP_BASIC_CONTENT_TYPE_FLAG) {
//        printf("Content-type: %.*s\n",
//               (int)envelope.message.properties.content_type.len,
//               (char *)envelope.message.properties.content_type.bytes);
//      }
///*      printf("----\n");*/

//      amqp_dump(envelope.message.body.bytes, envelope.message.body.len);
//	  begin_time = get_time_point();
//	  fprintf(fp, "%.*s  %lf \n", (char *)envelope.message.body.bytes,  begin_time/1e6);
//	      printf("rrt start time: %lf", begin_time/1e6);
//	      printf("----\n");
//      amqp_destroy_envelope(&envelope);

///////////////////////////////////////////////

//amqp_maybe_release_buffers(conn);
//    ret = amqp_consume_message(conn, &envelope, NULL, 0);
//      amqp_dump(envelope.message.body.bytes, envelope.message.body.len);
//	  begin_time = get_time_point();
//	  fprintf(fp, "%.*s  %lf ", (char *)envelope.message.body.bytes,  begin_time/1e6);
//	      printf("rrt start time: %lf", begin_time/1e6);
//	      printf("----\n");

//    if (AMQP_RESPONSE_NORMAL != ret.reply_type) {
//      if (AMQP_RESPONSE_LIBRARY_EXCEPTION == ret.reply_type &&
//          AMQP_STATUS_UNEXPECTED_STATE == ret.library_error) {
//        if (AMQP_STATUS_OK != amqp_simple_wait_frame(conn, &frame)) {
//          continue;
//        }

//        if (AMQP_FRAME_METHOD == frame.frame_type) {
//          switch (frame.payload.method.id) {
//            case AMQP_BASIC_ACK_METHOD:
//              /* if we've turned publisher confirms on, and we've published a
//               * message here is a message being confirmed.
//               */
//              break;
//            case AMQP_BASIC_RETURN_METHOD:
//              /* if a published message couldn't be routed and the mandatory
//               * flag was set this is what would be returned. The message then
//               * needs to be read.
//               */
//              {
//                amqp_message_t message;
//                ret = amqp_read_message(conn, frame.channel, &message, 0);


//                if (AMQP_RESPONSE_NORMAL != ret.reply_type) {
//                  continue;
//                }

//                amqp_destroy_message(&message);
//              }

//              break;

//            case AMQP_CHANNEL_CLOSE_METHOD:
//              /* a channel.close method happens when a channel exception occurs,
//               * this can happen by publishing to an exchange that doesn't exist
//               * for example.
//               *
//               * In this case you would need to open another channel redeclare
//               * any queues that were declared auto-delete, and restart any
//               * consumers that were attached to the previous channel.
//               */
//              continue;

//            case AMQP_CONNECTION_CLOSE_METHOD:
//              /* a connection.close method happens when a connection exception
//               * occurs, this can happen by trying to use a channel that isn't
//               * open for example.
//               *
//               * In this case the whole connection must be restarted.
//               */
//              continue;

//            default:
//              fprintf(stderr, "An unexpected method was received %u\n",
//                      frame.payload.method.id);
//              continue;
//          }
//        }
//      }

//    } else {
//      amqp_destroy_envelope(&envelope);
//    }

//////////////////////////////////////////////////////////



//    	double num_double = std::stod(s1);
	  double begin_time = get_time_point();
	  printf(" %lf \n", begin_time/1e6);
	  printf("GPU**===-------------------------------------------------===**\n");
	  printf("loopcount = %d \n",loopcount);
	  rrt.exploreCuda();

	  double time_diff = ((double)get_time_point() - begin_time) / 1e6;
	//fprintf(fp, " %lf \n", time_diff);
	  total_time+= time_diff;


	rsend(begin_time, time_diff);
//	  int sleep_val = 2.0-time_diff;
//	  if(sleep_val > 0)
//		usleep(sleep_val*1e6);
//	  	 else
//	{
//		printf("Path Planning Deadline missed \n");		
//		path_deadline_missed++;
//	}
 	  //printf("GPU Path finding Completed in %lf milli-seconds.\n\n\n", ((double)get_time_point() - begin_time) / 1000);

    }


//    printf("Average Time for Path Planning is %lf milli-seconds \n", total_time/(loopcount));
//   printf("Total Deadline Missed in Path Planning = %d\n",path_deadline_missed);
//	  #ifdef WR2F
//	  myfile1.close();
//	  #endif

	rend();


//	fclose(fp);
//  amqp_bytes_free(queuename);

//  die_on_amqp_error(amqp_channel_close(conn, 1, AMQP_REPLY_SUCCESS),
//                    "Closing channel");
//  die_on_amqp_error(amqp_connection_close(conn, AMQP_REPLY_SUCCESS),
//                    "Closing connection");
//  die_on_error(amqp_destroy_connection(conn), "Ending connection");

   destroy_cuda_mem();
   Destroy_stream();
return 0;

}









// end file

