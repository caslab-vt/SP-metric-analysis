#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <cassert>
#include <vector>
#include <ctime>
#include <chrono>
#include <math.h>
#include <unistd.h>

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <cmath>
#include <fstream>
#include <limits>
#include "rrt_memcpy.hpp"
#include <rabbitmq-c/amqp.h>
#include <rabbitmq-c/tcp_socket.h>
#include <string.h>
#include "utils.h"

//utils.c
#include <ctype.h>
#include <stdarg.h>
#include <rabbitmq-c/framing.h>

void die(const char *fmt, ...) {
  va_list ap;
  va_start(ap, fmt);
  vfprintf(stderr, fmt, ap);
  va_end(ap);
  fprintf(stderr, "\n");
  exit(1);
}

void die_on_error(int x, char const *context) {
  if (x < 0) {
    fprintf(stderr, "%s: %s\n", context, amqp_error_string2(x));
    exit(1);
  }
}

void die_on_amqp_error(amqp_rpc_reply_t x, char const *context) {
  switch (x.reply_type) {
    case AMQP_RESPONSE_NORMAL:
      return;

    case AMQP_RESPONSE_NONE:
      fprintf(stderr, "%s: missing RPC reply type!\n", context);
      break;

    case AMQP_RESPONSE_LIBRARY_EXCEPTION:
      fprintf(stderr, "%s: %s\n", context, amqp_error_string2(x.library_error));
      break;

    case AMQP_RESPONSE_SERVER_EXCEPTION:
      switch (x.reply.id) {
        case AMQP_CONNECTION_CLOSE_METHOD: {
          amqp_connection_close_t *m =
              (amqp_connection_close_t *)x.reply.decoded;
          fprintf(stderr, "%s: server connection error %uh, message: %.*s\n",
                  context, m->reply_code, (int)m->reply_text.len,
                  (char *)m->reply_text.bytes);
          break;
        }
        case AMQP_CHANNEL_CLOSE_METHOD: {
          amqp_channel_close_t *m = (amqp_channel_close_t *)x.reply.decoded;
          fprintf(stderr, "%s: server channel error %uh, message: %.*s\n",
                  context, m->reply_code, (int)m->reply_text.len,
                  (char *)m->reply_text.bytes);
          break;
        }
        default:
          fprintf(stderr, "%s: unknown server error, method id 0x%08X\n",
                  context, x.reply.id);
          break;
      }
      break;
  }

  exit(1);
}

static void dump_row(long count, int numinrow, int *chs) {
  int i;

  printf("%08lX:", count - numinrow);

  if (numinrow > 0) {
    for (i = 0; i < numinrow; i++) {
      if (i == 8) {
        printf(" :");
      }
      printf(" %02X", chs[i]);
    }
    for (i = numinrow; i < 16; i++) {
      if (i == 8) {
        printf(" :");
      }
      printf("   ");
    }
    printf("  ");
    for (i = 0; i < numinrow; i++) {
      if (isprint(chs[i])) {
        printf("%c", chs[i]);
      } else {
        printf(".");
      }
    }
  }
  printf("\n");
}

static int rows_eq(int *a, int *b) {
  int i;

  for (i = 0; i < 16; i++)
    if (a[i] != b[i]) {
      return 0;
    }

  return 1;
}

void amqp_dump(void const *buffer, size_t len) {
  unsigned char *buf = (unsigned char *)buffer;
  long count = 0;
  int numinrow = 0;
  int chs[16];
  int oldchs[16] = {0};
  int showed_dots = 0;
  size_t i;

  for (i = 0; i < len; i++) {
    int ch = buf[i];

    if (numinrow == 16) {
      int j;

      if (rows_eq(oldchs, chs)) {
        if (!showed_dots) {
          showed_dots = 1;
          printf(
              "          .. .. .. .. .. .. .. .. : .. .. .. .. .. .. .. ..\n");
        }
      } else {
        showed_dots = 0;
        dump_row(count, numinrow, chs);
      }

      for (j = 0; j < 16; j++) {
        oldchs[j] = chs[j];
      }

      numinrow = 0;
    }

    count++;
    chs[numinrow++] = ch;
  }

  dump_row(count, numinrow, chs);

  if (numinrow != 0) {
    printf("%08lX:\n", count);
  }
}


//end utils.c functions

#define BLOCK 1024



static hipStream_t streamsArray[4];
static int streamInit[4] = { 0 };

hipStream_t get_cuda_stream_32(int i) {
    if (!streamInit[i]) {
	hipError_t status = hipStreamCreateWithFlags(&streamsArray[i],hipStreamNonBlocking);
        streamInit[i] = 1;
    }
    return streamsArray[i];
}

void Destroy_stream()
{
    for(int i =0;i<4;i++)
    {
	if (streamInit[i]) {
    		hipStreamDestroy(streamsArray[i]);
   		streamInit[i] = 0;
	}
    }
}

double get_time_point() {
    std::chrono::steady_clock::time_point current_time = std::chrono::steady_clock::now();
    return std::chrono::duration_cast<std::chrono::microseconds>(current_time.time_since_epoch()).count();
}


int get_number_of_blocks(int array_size, int block_size)
{
    return array_size / block_size + ((array_size % block_size > 0) ? 1 : 0);
}

__global__ void distance_cal_cuda(vertex* vertex_cuda_ptr,float* new_vertex_cuda,float* distances,int offset)
{
	int idx = blockIdx.x * blockDim.x + offset +  threadIdx.x;
	float current_vertex[2] = {vertex_cuda_ptr[idx].x,vertex_cuda_ptr[idx].y};

	float dx = current_vertex[0] - new_vertex_cuda[0];
	float dy = current_vertex[1] - new_vertex_cuda[1];
	
	distances[idx] = sqrt((dx*dx) + (dy * dy));

}

//whether the new edge collides with polygon
__device__ bool overlapping_check_dev(float p0, float p3, float p1, float p2)
{
	if (p0 > p3)
	{
	   p0 = p0+p3;
	   p3 = p0 - p3;
	   p0 = p0 - p3;
	}
	if (p1 > p2)
	{
	   p1 = p1+p2;
	   p2 = p1 - p2;
	   p1 = p1 - p2;
	}
	float max_p00_p10 = p0 > p1 ?p0 : p1;
	float min_p30_p20 = p3 < p2 ?p3 : p2;
	return max_p00_p10 <= min_p30_p20;
	
}

__global__ void polygon_collision_check_dev(Polygon* polygons_ ,float* p0,int offset)
{
	int i = blockIdx.x * blockDim.x + offset + threadIdx.x;
	
	__shared__ int flag;
	if(threadIdx.x == 0)
		flag =0;
	__syncthreads();
	
	if(flag == 1 || p0[4] == 1)
	{
		return;
	}
	float p1[2];
  	float p2[2];

	p1[0] = polygons_[i].x1;
	p1[1] = polygons_[i].y1;
  	p2[0] = polygons_[i].x1;
	p2[1] = polygons_[i].y2;

	float cross_p0_p1_p2 = (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
	float cross_p3_p1_p2 = (((p1[0] - p0[2]) * (p2[1] - p0[3])) - ((p2[0] - p0[2]) * (p1[1] - p0[3])));
	float cross_p1_p0_p3 = (((p0[0] - p1[0]) * (p0[3] - p1[1])) - ((p0[2] - p1[0]) * (p0[1] - p1[1])));
	float cross_p2_p0_p3 = (((p0[0] - p2[0]) * (p0[3] - p2[1])) - ((p0[2] - p2[0]) * (p0[1] - p2[1])));
	int sign_cross_p0_p1_p2 =  cross_p0_p1_p2 >=0 ? cross_p0_p1_p2 ? 1 :0 :-1;
	int sign_cross_p3_p1_p2 =  cross_p3_p1_p2 >=0 ? cross_p3_p1_p2 ? 1 :0 :-1;
	int sign_cross_p1_p0_p3 =  cross_p1_p0_p3 >=0 ? cross_p1_p0_p3 ? 1 :0 :-1;
	int sign_cross_p2_p0_p3 =  cross_p2_p0_p3 >=0 ? cross_p2_p0_p3 ? 1 :0 :-1;	

	bool overlapping_check_x = overlapping_check_dev(p0[0],p0[2],p1[0],p2[0]);
	bool overlapping_check_y = overlapping_check_dev(p0[1],p0[3],p1[1],p2[1]);
	
	if((cross_p0_p1_p2 == 0) &&  (cross_p3_p1_p2 == 0)) // Lines are colinear
	{
		if (overlapping_check_x || overlapping_check_y) //lines are overlapping
		{
			flag = 1;
			p0[4] = 1;
			return;	
		}
			
	}
	
	if((sign_cross_p0_p1_p2 != sign_cross_p3_p1_p2) && (sign_cross_p1_p0_p3 != sign_cross_p2_p0_p3))    //lines are intersecting
	{
		flag = 1;
		p0[4] = 1;
		return;	
	}
	if(flag == 1 || p0[4] == 1)
	{
		return;
	}
	p1[0] = polygons_[i].x1;
	p1[1] = polygons_[i].y2;
  	p2[0] = polygons_[i].x2;
	p2[1] = polygons_[i].y2;

	cross_p0_p1_p2 = (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
	cross_p3_p1_p2 = (((p1[0] - p0[2]) * (p2[1] - p0[3])) - ((p2[0] - p0[2]) * (p1[1] - p0[3])));
	cross_p1_p0_p3 = (((p0[0] - p1[0]) * (p0[3] - p1[1])) - ((p0[2] - p1[0]) * (p0[1] - p1[1])));
	cross_p2_p0_p3 = (((p0[0] - p2[0]) * (p0[3] - p2[1])) - ((p0[2] - p2[0]) * (p0[1] - p2[1])));
	sign_cross_p0_p1_p2 =  cross_p0_p1_p2 >=0 ? cross_p0_p1_p2 ? 1 :0 :-1;
	sign_cross_p3_p1_p2 =  cross_p3_p1_p2 >=0 ? cross_p3_p1_p2 ? 1 :0 :-1;
	sign_cross_p1_p0_p3 =  cross_p1_p0_p3 >=0 ? cross_p1_p0_p3 ? 1 :0 :-1;
	sign_cross_p2_p0_p3 =  cross_p2_p0_p3 >=0 ? cross_p2_p0_p3 ? 1 :0 :-1;	

	overlapping_check_x = overlapping_check_dev(p0[0],p0[2],p1[0],p2[0]);
	overlapping_check_y = overlapping_check_dev(p0[1],p0[3],p1[1],p2[1]);
		
	if((cross_p0_p1_p2 == 0) &&  (cross_p3_p1_p2 == 0)) // Lines are colinear
	{
		if (overlapping_check_x || overlapping_check_y) //lines are overlapping
		{
			flag = 1;
			p0[4] = 1;
			return;	
		}
			
	}
	
	if((sign_cross_p0_p1_p2 != sign_cross_p3_p1_p2) && (sign_cross_p1_p0_p3 != sign_cross_p2_p0_p3))    //lines are intersecting
	{
		flag = 1;
		p0[4] = 1;
		return;	
	}
	
	if(flag == 1 || p0[4] == 1)
	{
		return;
	}
	p1[0] = polygons_[i].x2;
	p1[1] = polygons_[i].y2;
  	p2[0] = polygons_[i].x2;
	p2[1] = polygons_[i].y1;
	
	cross_p0_p1_p2 = (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
	cross_p3_p1_p2 = (((p1[0] - p0[2]) * (p2[1] - p0[3])) - ((p2[0] - p0[2]) * (p1[1] - p0[3])));
	cross_p1_p0_p3 = (((p0[0] - p1[0]) * (p0[3] - p1[1])) - ((p0[2] - p1[0]) * (p0[1] - p1[1])));
	cross_p2_p0_p3 = (((p0[0] - p2[0]) * (p0[3] - p2[1])) - ((p0[2] - p2[0]) * (p0[1] - p2[1])));
	sign_cross_p0_p1_p2 =  cross_p0_p1_p2 >=0 ? cross_p0_p1_p2 ? 1 :0 :-1;
	sign_cross_p3_p1_p2 =  cross_p3_p1_p2 >=0 ? cross_p3_p1_p2 ? 1 :0 :-1;
	sign_cross_p1_p0_p3 =  cross_p1_p0_p3 >=0 ? cross_p1_p0_p3 ? 1 :0 :-1;
	sign_cross_p2_p0_p3 =  cross_p2_p0_p3 >=0 ? cross_p2_p0_p3 ? 1 :0 :-1;	

	overlapping_check_x = overlapping_check_dev(p0[0],p0[2],p1[0],p2[0]);
	overlapping_check_y = overlapping_check_dev(p0[1],p0[3],p1[1],p2[1]);
		
	if((cross_p0_p1_p2 == 0) &&  (cross_p3_p1_p2 == 0)) // Lines are colinear
	{
		if (overlapping_check_x || overlapping_check_y) //lines are overlapping
		{
			flag = 1;
			p0[4] = 1;
			return;	
		}
			
	}
	
	if((sign_cross_p0_p1_p2 != sign_cross_p3_p1_p2) && (sign_cross_p1_p0_p3 != sign_cross_p2_p0_p3))    //lines are intersecting
	{
		flag = 1;
		p0[4] = 1;
		return;	
	}	
	if(flag == 1 || p0[4] == 1)
	{
		return;
	}
	p1[0] = polygons_[i].x2;
	p1[1] = polygons_[i].y1;
  	p2[0] = polygons_[i].x1;
	p2[1] = polygons_[i].y1;
	
	cross_p0_p1_p2 = (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
	cross_p3_p1_p2 = (((p1[0] - p0[2]) * (p2[1] - p0[3])) - ((p2[0] - p0[2]) * (p1[1] - p0[3])));
	cross_p1_p0_p3 = (((p0[0] - p1[0]) * (p0[3] - p1[1])) - ((p0[2] - p1[0]) * (p0[1] - p1[1])));
	cross_p2_p0_p3 = (((p0[0] - p2[0]) * (p0[3] - p2[1])) - ((p0[2] - p2[0]) * (p0[1] - p2[1])));
	sign_cross_p0_p1_p2 =  cross_p0_p1_p2 >=0 ? cross_p0_p1_p2 ? 1 :0 :-1;
	sign_cross_p3_p1_p2 =  cross_p3_p1_p2 >=0 ? cross_p3_p1_p2 ? 1 :0 :-1;
	sign_cross_p1_p0_p3 =  cross_p1_p0_p3 >=0 ? cross_p1_p0_p3 ? 1 :0 :-1;
	sign_cross_p2_p0_p3 =  cross_p2_p0_p3 >=0 ? cross_p2_p0_p3 ? 1 :0 :-1;	
	overlapping_check_x = overlapping_check_dev(p0[0],p0[2],p1[0],p2[0]);
	overlapping_check_y = overlapping_check_dev(p0[1],p0[3],p1[1],p2[1]);
		
	if((cross_p0_p1_p2 == 0) &&  (cross_p3_p1_p2 == 0)) // Lines are colinear
	{
		if (overlapping_check_x || overlapping_check_y) //lines are overlapping
		{
			flag = 1;
			p0[4] = 1;
			return;	
		}
			
	}
	
	if((sign_cross_p0_p1_p2 != sign_cross_p3_p1_p2) && (sign_cross_p1_p0_p3 != sign_cross_p2_p0_p3))    //lines are intersecting
	{
		flag = 1;
		p0[4] = 1;
		return;	
	}
	
}

float distance_(const float *p1, const float *p2)
{
  const float dx = p1[0] - p2[0];
  const float dy = p1[1] - p2[1];
  

  return sqrt(pow(dx, 2) + pow(dy, 2));
}

int min_distance(float* distance_list,int vertex_count)
{
	float min_dstance = distance_list[0];
	int idx = 0;	
	for(int i=1; i <vertex_count;i++)
	{
		if(distance_list[i] < min_dstance)
		{
			min_dstance = distance_list[i];
			idx = i;
		} 
	}

	return idx;
}


float cross_(const float *p0, const float *p1, const float *p2)
{
	return (((p1[0] - p0[0]) * (p2[1] - p0[1])) - ((p2[0] - p0[0]) * (p1[1] - p0[1])));
}

int sign_(float value)
{
	return value >=0 ? value ? 1 :0 :-1;
}

float max_(float p0 , float p1)
{
	return p0 > p1?p0 : p1;
}

float min_(float p0 , float p1)
{
	return p0 < p1?p0 : p1;
}


bool overlapping_check(float p0, float p3, float p1, float p2)
{
	if (p0 > p3)
	{
	   p0 = p0+p3;
	   p3 = p0 - p3;
	   p0 = p0 - p3;
	}
	if (p1 > p2)
	{
	   p1 = p1+p2;
	   p2 = p1 - p2;
	   p1 = p1 - p2;
	}
	return max_(p0,p1) <= min_(p3,p2);
	
}

RRT::RRT(float *start, float *goal, int rando)
        : start_(start),
          goal_(goal),
          delta_(0.1),
          epsilon_(0),
          xmin_(0),
          xmax_(1000),
          ymin_(0),
          ymax_(1000),
          resolution_(1.0),
          max_iter_(MAX_ITER),
          vertex_count_(0),
	  obstacles_count_(0)
{
  std::srand(rando);
}

bool RRT::polygon_collision_check_cuda(const vertex &v_new,const vertex &v_near,int goal_check)
{
	#ifdef WR2F
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	#endif	
	static double total_time1 = 0;
  	static int count1 = 0;
	double begin_time = get_time_point();
	v_new_near_host[0] = v_new.x;
	v_new_near_host[1] = v_new.y;	
	v_new_near_host[2] = v_near.x;
	v_new_near_host[3] = v_near.y;
	v_new_near_host[4] = 0;	
	
	////////////////////////////////////////////////////////////////
	int num_blocks = get_number_of_blocks(obstacles_count_, BLOCK);
	int extra_blocks = num_blocks %4;
	int stream_id = 0;
	num_blocks = num_blocks /4;
	int loopcount = 0;
	int offset = 0;
	for (loopcount = 0; loopcount < num_blocks;loopcount++)
	{
		offset = loopcount * 4*BLOCK;
		stream_id = loopcount%4;
		polygon_collision_check_dev<<<4, BLOCK, 0, get_cuda_stream_32(stream_id)>>>(polygons_dev,v_new_near_dev,offset);	
	}
	if(extra_blocks)
	{
		stream_id = loopcount%4;
		offset = loopcount * 4*BLOCK;
		polygon_collision_check_dev<<<extra_blocks, BLOCK, 0, get_cuda_stream_32(stream_id)>>>(polygons_dev,v_new_near_dev,offset);	
	}
	if(loopcount == 0)
	{
		hipEventRecord(stopEvent,get_cuda_stream_32(0));
		while(hipEventQuery(stopEvent) != 0);
	}
	else
	{
		hipDeviceSynchronize();
	}
	////////////////////////////////////////////////////////////////

	/*int blocksize_ = obstacles_count_ > 1024?1024:obstacles_count_ ;
	int gridsize_ = obstacles_count_ % 1024?int(obstacles_count_/1024) + 1:int(obstacles_count_/1024);
        int offset = 0;	*/
	#ifdef WR2F
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	duration<double, milli> time_span = t2-t1;
	myfile1 <<"Obstacle Detection = "  << time_span.count() << endl; 
	#endif	
	count1++;
	double time_diff = ((double)get_time_point() - begin_time) / 1000;
	total_time1+= time_diff;
	if(v_new_near_host[4])
	{	
		return true;
	}
	else
	{
		if(goal_check)
			printf("Average Time for collision check is %lf milli-seconds \n", total_time1/(count1));
		return false;
	}
}


bool RRT::PolygonCollisionCheck(const vertex &v_new, const vertex &v_near)
{

  
  const float p0[2] = {v_new.x, v_new.y};
  const float p3[2] = {v_near.x, v_near.y};
  
  float p1[2];
  float p2[2];

  for(unsigned int i = 0; i < obstacles_count_; i++)
  {
	p1[0] = polygons_[i].x1;
	p1[1] = polygons_[i].y1;
  	p2[0] = polygons_[i].x1;
	p2[1] = polygons_[i].y2;
	
	if((cross_(p0, p1, p2) == 0) &&  (cross_(p3, p1, p2) == 0)) // Lines are colinear
	{
		if (overlapping_check(p0[0], p3[0], p1[0], p2[0]) || overlapping_check(p0[1], p3[1], p1[1], p2[1])) //lines are overlapping
		{
			return true; 	
		}
			
	}
	
	if((sign_(cross_(p0, p1, p2)) != sign_(cross_(p3, p1, p2))) && (sign_(cross_(p1, p0, p3)) != sign_(cross_(p2, p0, p3))))    //lines are intersecting
	{

			return true;
	}
	
	/*if ((i==1249))
	{
		printf(" CPU %d - %f ,%f , %f, %f \n",i,cross_(p0, p1, p2),cross_(p3, p1, p2),cross_(p1, p0, p3),cross_(p2, p0, p3));
		printf("CPU - %d, p1 = %f %f, p2 = %f %f, p0 = %f %f, p3 = %f %f \n ",i,p1[0],p1[1],p2[0],p2[1],p0[0],p0[1],p3[0],p3[1]);
	}*/
		
	
	p1[0] = polygons_[i].x1;
	p1[1] = polygons_[i].y2;
  	p2[0] = polygons_[i].x2;
	p2[1] = polygons_[i].y2;
	
	if((cross_(p0, p1, p2) == 0) &&  (cross_(p3, p1, p2) == 0)) // Lines are colinear
	{
		if (overlapping_check(p0[0], p3[0], p1[0], p2[0]) || overlapping_check(p0[1], p3[1], p1[1], p2[1])) //lines are overlapping
		{
			return true; 	
		}
			
	}
	
	if((sign_(cross_(p0, p1, p2)) != sign_(cross_(p3, p1, p2))) && (sign_(cross_(p1, p0, p3)) != sign_(cross_(p2, p0, p3))))    //lines are intersecting
	{
			return true;
		
	}	
	
	p1[0] = polygons_[i].x2;
	p1[1] = polygons_[i].y2;
  	p2[0] = polygons_[i].x2;
	p2[1] = polygons_[i].y1;
	
	if((cross_(p0, p1, p2) == 0) &&  (cross_(p3, p1, p2) == 0)) // Lines are colinear
	{
		if (overlapping_check(p0[0], p3[0], p1[0], p2[0]) || overlapping_check(p0[1], p3[1], p1[1], p2[1])) //lines are overlapping
		{
			return true; 	
		}
			
	}
	
	if((sign_(cross_(p0, p1, p2)) != sign_(cross_(p3, p1, p2))) && (sign_(cross_(p1, p0, p3)) != sign_(cross_(p2, p0, p3))))    //lines are intersecting
	{
			return true;
		
	}

	p1[0] = polygons_[i].x2;
	p1[1] = polygons_[i].y1;
  	p2[0] = polygons_[i].x1;
	p2[1] = polygons_[i].y1;
	
	if((cross_(p0, p1, p2) == 0) &&  (cross_(p3, p1, p2) == 0)) // Lines are colinear
	{
		if (overlapping_check(p0[0], p3[0], p1[0], p2[0]) || overlapping_check(p0[1], p3[1], p1[1], p2[1])) //lines are overlapping
		{
			return true;	
		}
			
	}
	
	if((sign_(cross_(p0, p1, p2)) != sign_(cross_(p3, p1, p2))) && (sign_(cross_(p1, p0, p3)) != sign_(cross_(p2, p0, p3))))    //lines are intersecting
	{
			return true;
		
	}
  }
  return false;
}

bool RRT::exploreObstacles()
{

  vertex_count_ = 0;
  //vertices_.clear();
  // add start to graph
  vertex v_start;
  v_start.x = start_[0];
  v_start.y = start_[1];
  addVertex(v_start);


  bool success = false;
  int ctr = 0;


  while(!success)
  {
	
    if (ctr > max_iter_)
    {
      std::cout << "Goal not achieved" << std::endl;
      return false;
    }

    // std::cout << "Iter: " << ctr << std::endl;
	float rand[2];
    // 1) random point
    randomConfig(rand);
// std::cout << "randat: " << rand_point_cuda[0] << ", " << rand_point_cuda[1] <<std::endl;

    // 2) nearest node in graph
    vertex v_near;
    high_resolution_clock::time_point t3 = high_resolution_clock::now();
     nearestVertex(v_near, rand);
    high_resolution_clock::time_point t4 = high_resolution_clock::now();
    duration<double, milli> time_span = t4-t3;
    #ifdef WR2F
    myfile <<"Nearest Vertex = "  << time_span.count() << endl; 
    #endif 
    // 3) new node
    vertex v_new;
    if(!newConfiguration(v_new, v_near, rand))
    {
      continue;
    }

//    std::cout << "v new at: " << v_new.x << ", " << v_new.y <<std::endl;

    ctr++;

    // 4) check for collisions
   // if (collision_check(v_new, v_near))
   // {
      // std::cout << "Collision" << std::endl;
   //   continue;
   // }
   t3 = high_resolution_clock::now();
   bool collision_flag = PolygonCollisionCheck(v_new, v_near);
   t4 = high_resolution_clock::now();
   time_span = t4-t3;
    #ifdef WR2F
    myfile <<"Obstacle Detection = "  << time_span.count() << endl;  
   #endif
   //cout <<  "CPU Time " << time_span.count() << " ms" << endl; 
    if (collision_flag)
    {
	//std::cout << "Collision" << std::endl;
       continue;
    }

    // std::cout << v_new.x << " " << v_new.y << "\n";

    // 6) add new node
    addVertex(v_new);
    addEdge(v_near, v_new);

    // 7) win check
  t3 = high_resolution_clock::now();
  bool win_flag = win_check(v_new, goal_);
  t4 = high_resolution_clock::now();
   time_span = t4-t3;
   #ifdef WR2F
   myfile <<"Obstacle Detection = "  << time_span.count() << endl; 
   #endif
  // cout <<  "CPU Time " << time_span.count() << " ms" << endl; 
  //  win_flag = 0;
    if (win_flag)
    {
      std::cout << "Goal reached on CPU" << std::endl;
      printf("ctr = %d \n",ctr);
      // add goal to graph
      vertex v_goal;
      v_goal.x = goal_[0];
      v_goal.y = goal_[1];
      addVertex(v_goal);
      addEdge(v_new, v_goal);
      std::cout << "Vnew " << v_new.x << " " << v_new.y << "\n";
      std::cout << "Goal " << v_goal.x << " " << v_goal.y << "\n";
      success = true;
      break;
    }

  }

  return success;
}


bool RRT::win_check(const vertex &v_new, const float *goal)
{
  //cast goal to vertex //TODO: overlead collision to optionally take float as second arg
  vertex v_goal(goal[0],goal[1]);
  // std::cout << "SURUR\n";
  bool collis_check = PolygonCollisionCheck(v_new, v_goal);

 // std::cout << collis_check << std::endl;

  return !collis_check;
}

bool RRT::win_check_cuda(const vertex &v_new,const float *goal)
{
  vertex v_goal(goal[0],goal[1]);
  bool collis_check = polygon_collision_check_cuda(v_new,v_goal,1);
//PolygonCollisionCheck(v_new, v_goal,polygons_dev);

 // std::cout << collis_check << std::endl;

  return !collis_check;
}

bool RRT::exploreCuda()
{
  static double total_time = 0;
  static int count = 0;
  double time_diff;
   vertex_count_ = 0;
 // vertices_.clear();
   // add start to graph
  vertex v_start;
  v_start.x = start_[0];
  v_start.y = start_[1];
  addVertex(v_start);
  bool success = false;
  int ctr = 0;
  while(!success)
  {
	
    if (ctr > max_iter_)
    {
      std::cout << "Goal not achieved" << std::endl;
      return false;
    }
    // printf("ctr = %d\n",ctr);
    float rand[2];
    randomConfig(rand); //generate the random point
    vertex v_near;	
 //   high_resolution_clock::time_point t3 = high_resolution_clock::now();
    double begin_time = get_time_point();
    if(vertex_count_ > 1024)
    	nearestVertex_cuda(v_near, rand); //use cuda to calculate the distances betwen the random generated point and all the vertexes
    else
     	nearestVertex(v_near, rand);	
//returns vnear
    count++;
    time_diff = ((double)get_time_point() - begin_time) / 1000;
    total_time+= time_diff;
   // high_resolution_clock::time_point t4 = high_resolution_clock::now();
  //  duration<double, milli> time_span = t4-t3;
    #ifdef WR2F
    myfile1 <<"Nearest Vertex = "  << time_span.count() << endl; 
    #endif
    vertex v_new;
	
    if(!newConfiguration(v_new, v_near, rand))
    {
	
      continue;
    }
    ctr++;
	
    if (polygon_collision_check_cuda(v_new,v_near,0)) //check if new edge is colliding with any polygon
    {
       continue;
    }
    addVertex(v_new); //
    addEdge(v_near, v_new);

    bool win_flag = win_check_cuda(v_new,goal_); //can we draw directly line to goal
    if (win_flag)
    {
	printf("Average Time for Nearest Vertex find is %lf milli-seconds \n", total_time/(count));
     // std::cout << "Goal reached on GPU" << std::endl;
      printf("ctr = %d \n",ctr);
      vertex v_goal;
      v_goal.x = goal_[0];
      v_goal.y = goal_[1];
      addVertex(v_goal);
      addEdge(v_new, v_goal);
     // std::cout << "Vnew " << v_new.x << " " << v_new.y << "\n";
    //  std::cout << "Goal " << v_goal.x << " " << v_goal.y << "\n";
      success = true;
      break;
    }

  }

  return success;
}

//here 3d
void RRT::randomObstacles(int num_obstacle, float x_min, float x_max, float y_min, float y_max)
{
	for(int i = 0; i<num_obstacle;i++)
	{
		 float x1 = xmin_+static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(xmax_-xmin_)));
    		 float y1 = ymin_+static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(ymax_-ymin_)));
		 float x2 = x1 + x_min +static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(x_max-x_min)));
    		 float y2 = y1 +y_min + +static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(y_max- y_min)));

		if ((start_[0] >= x1) && (start_[0] <= x2) && (start_[1] >= y1) && (start_[1] <= y2))
		{
			i--;
			//printf("%f %f %f %f start within the polygon\n ",x1,y1,x2,y2);
			continue;
		}
		if (goal_[0] >= x1 && goal_[0] <= x2 && goal_[1] >= y1 && goal_[1] <= y2)
		{
			i--;
			//printf("%f %f %f %f goal within the polygon\n ",x1,y1,x2,y2);	
			continue;
		}
		Polygon poly;
		poly.x1 = x1;
		poly.y1 = y1;
		poly.x2 = x2;	
		poly.y2 = y2;
		//polygons_.push_back(poly);
		//polygons_dev[i] = poly;
		polygons_[obstacles_count_++] = poly;
		//printf(" polygo= %f %f %f %f \n",x1,y1,x2,y2);
		//calculate obstacle density here
		
	}
	//printf("size = %d\n",polygons_.size());
}


//void RRT::traverseGraph(std::vector<vertex> &path) const
//{
//  // path.reserve(vertices_.size());
//  std::ofstream pathout;
//  pathout.open("rrtout/path.csv");

//  int start_idx = 0;                  // first vertex added
//  int goal_idx = vertex_count_-1;  // last vertex added


//  // std::cout << "start: " << start_idx << std::endl;
//  // std::cout << "goal: " << goal_idx << std::endl;


//  // path is backwards
//  path.push_back(vertices_.at(goal_idx));

//  // current vertex is the goal
//  //vertex curr_v = vertices_.at(goal_idx);
//  vertex curr_v = vertex_host_ptr[goal_idx];
//  int curr_idx = goal_idx;


//  while(curr_idx != start_idx)
//  {

//    int parent_idx = findParent(curr_v);
//    pathout << vertices_.at(curr_idx).x << "," << vertices_.at(curr_idx).y << "," << vertices_.at(curr_idx).z << "," 
//            << vertices_.at(parent_idx).x << "," << vertices_.at(parent_idx).y << "," << vertices_.at(parent_idx).z << "\n";

//    path.push_back(vertices_.at(parent_idx));

//    // update current node and current index
//    curr_v = vertices_.at(parent_idx);
//    curr_idx = parent_idx;
//  }
//}



//void RRT::printGraph() const
//{
//  for(unsigned int i = 0; i < vertices_.size(); i++)
//  {
//    std::cout << "vertex: " << vertices_.at(i).id << " -> ";
//    for(unsigned int j = 0; j < vertices_.at(i).adjacent_vertices.size(); j++)
//    {
//      std::cout << vertices_.at(i).adjacent_vertices.at(j) << " ";
//    }
//    std::cout << std::endl;
//  }
//}



void RRT::addVertex(vertex &v)
{
  v.id = vertex_count_;

  vertex_host_ptr[vertex_count_++] = v;
//
  

  // std::cout << "New vertex count: " << vertex_count_ << std::endl;
} 



void RRT::addEdge(const vertex &v_near, const vertex &v_new)
{
  // search for node1 and node2
  // addes edge btw both
  bool added = false;


  for(unsigned int i = 0; i < vertex_count_; i++)
  {
    // found node 1
    if (vertex_host_ptr[i].id == v_near.id)
    {
      for(unsigned int j = 0; j < vertex_count_; j++)
      {
        // do not add vertex to itself
        // found node 2
        if(vertex_host_ptr[j].id == v_new.id && i != j)
        {
          // edge connecting node 1 to node 2
          // std::cout << "adding edge " << v_near.id << "->" << v_new.id << std::endl;
          // v_near.adjacent_vertices.push_back(v_new.id);
	  //v_new.rank=v_near.rank+1;
          vertex_host_ptr[v_near.id].adjacent_vertices.push_back(v_new.id);
          added = true;
        }

      } // end inner loop
    }
  } // end outer loop

  if (!added)
  {
    std::cout << "Error: 'addEdge' edge not added" << std::endl;
  }
}


bool RRT::newConfiguration(vertex &v_new, const vertex &v_near, const float *q_rand) const
{

  //std::cout << "Vnear " << v_near.x << " " << v_near.y << "\n";
  // difference btw q_rand and v_near
  const float vx = q_rand[0] - v_near.x;
  const float vy = q_rand[1] - v_near.y;
  
//std::cout << "Vx " << vx << " " << vy << "\n";
  // distance_ between v_near and q_rand
  const float magnitude = std::sqrt(std::pow(vx, 2) + std::pow(vy, 2));
  
//  std::cout << "Magnitude " << magnitude << std::endl;

  if (magnitude == 0)
  {
    return false;
  }

  // unit vector in driection of q_rand
  const float ux = vx / magnitude;
  const float uy = vy / magnitude;

  // place v_new a delta away from v_near
  v_new.x = v_near.x + delta_ * ux;
  v_new.y = v_near.y + delta_ * uy;
 //std::cout << "ux " << ux << " " << uy << "\n";
 // std::cout << "Vnew " << v_new.x << " " << v_new.y << "\n";

  // make sure still within bounds
  if (v_new.x > xmax_ || v_new.x < xmin_ || v_new.y > ymax_ || v_new.y < ymin_)
  {
    return false;
  }

  return true;
}



void RRT::nearestVertex(vertex &v, float *q_rand) const
{
  float point[2];
  std::vector<float> d;

  for(unsigned int i = 0; i < vertex_count_; i++)
  {
    point[0] = vertex_host_ptr[i].x;
    point[1] = vertex_host_ptr[i].y;

    distances_host[i] = (distance_(point, q_rand));
  }

  int idx = min_distance(distances_host,vertex_count_);
  v = vertex_host_ptr[idx];
}

void RRT::nearestVertex_cuda(vertex &v, float *q_rand) const
{
	
	/*int blocksize_ = vertex_count_ > 1024?1024:vertex_count_ ;
	int gridsize_ = vertex_count_ % 1024?int(vertex_count_/1024) + 1:int(vertex_count_/1024);
	dim3 blocksize(blocksize_);
	dim3 gridsize(gridsize_);*/
	new_vertex_host[0] = q_rand[0];
	new_vertex_host[1] = q_rand[1];
	//////////////////////////////////////////////////////////////////////////////
	int num_blocks = get_number_of_blocks(vertex_count_, BLOCK);
	int extra_blocks = num_blocks %4;
	int stream_id = 0;
	num_blocks = num_blocks /4;
	int loopcount = 0;
	int offset = 0;
	for (loopcount = 0; loopcount < num_blocks;loopcount++)
	{
		offset = loopcount * 4*BLOCK;
		stream_id = loopcount%4;
		distance_cal_cuda<<<4, BLOCK, 0, get_cuda_stream_32(stream_id)>>>(vertex_cuda_ptr,new_vertex_cuda,distances_cuda,offset);	
	}
	if(extra_blocks)
	{
		stream_id = loopcount%4;
		offset = loopcount * 4*BLOCK;
		distance_cal_cuda<<<extra_blocks, BLOCK, 0, get_cuda_stream_32(stream_id)>>>(vertex_cuda_ptr,new_vertex_cuda,distances_cuda,offset);	
	}
	if(loopcount == 0)
	{
		hipEventRecord(stopEvent,get_cuda_stream_32(0));
		while(hipEventQuery(stopEvent) != 0);
	}
	else
	{
		hipDeviceSynchronize();
	}
	////////////////////////////////////////////////////////////////////////////////


	/*distance_cal_cuda<<<gridsize,blocksize,0,get_cuda_stream_32(0)>>>(vertex_cuda_ptr,new_vertex_cuda,distances_cuda,offset);
	hipEventRecord(stopEvent,stream[0]);
	while(hipEventQuery(stopEvent) != 0);*/
	int idx = min_distance(distances_host,vertex_count_);
	v = vertex_host_ptr[idx];
	
}


void RRT::randomConfig(float *q_rand) const
{
  // x position
  q_rand[0] = xmin_+static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(xmax_-xmin_)));

  // y position
  q_rand[1] = ymin_+static_cast<float>(std::rand()) / (static_cast<float>(RAND_MAX/(ymax_-ymin_)));
}

int RRT::findParent(const vertex &v) const
{
  // iterate over vertices
  for(unsigned int i = 0; i < vertex_count_; i++)
  {
    for(unsigned int j = 0; j < vertex_host_ptr[i].adjacent_vertices.size(); j++)
    {
      if (vertex_host_ptr[i].adjacent_vertices.at(j) == v.id)
      {
        // std::cout << "Parent found" << std::endl;
        return i;
      }
    } // end inner loop
  } // end outer loop

  std::cout << "Parent not found" << std::endl;
  return -1;
}

void destroy_cuda_mem(void)
{
 // hipFree(polygons_dev);
  hipFree(v_new_near_dev);
  hipHostFree(polygons_);
  hipHostFree(v_new_near_host);
  hipFree(vertex_cuda_ptr);
  hipFree(new_vertex_cuda);
  hipFree(distances_cuda);
  hipHostFree(vertex_host_ptr);
  hipHostFree(new_vertex_host);
  hipHostFree(distances_host);
  hipEventDestroy(stopEvent);

}


int main(int argc, char * argv[])
{

 ///  cpu_set_t cpuset_sched;
//CPU_ZERO(&cpuset_sched);
//CPU_SET(4,&cpuset_sched);
//sched_setaffinity(getpid(),sizeof(cpuset_sched),&cpuset_sched);

  printf("\n\n**===-------------------------------------------------===**\n");
  printf("\n\n**===--- RRT ---===**\n");
  printf("\n\n**===-------------------------------------------------===**\n");
  float start[] = {0,0};
  float goal[] = {1000,1000};//{float(rand()%1000),float(rand()%1000)};
  

  //expecting command line args to be rand num, then num onbstacles
  if (argc > 2)
  {
    num_obstacles = std::atoi(argv[2]);
    rand_num = std::atoi(argv[1]);
  } else if (argc > 1) {
    rand_num = std::atoi(argv[1]);
    num_obstacles = 2048;
  } else {
    num_obstacles = 2048;
    rand_num = 20;
  }

  RRT rrt(start, goal, rand_num);
  hipHostAlloc(&polygons_,num_obstacles*sizeof(Polygon),hipHostMallocMapped);
  rrt.randomObstacles(num_obstacles, 1.0, 2.0, 1.0, 2.0);
  hipHostAlloc(&v_new_near_host,5*sizeof(float),hipHostMallocMapped);
  status = hipHostGetDevicePointer(&v_new_near_dev,v_new_near_host,0);  
  hipHostAlloc(&vertex_host_ptr,MAX_ITER*sizeof(vertex),hipHostMallocMapped);
  hipHostAlloc(&new_vertex_host,2*sizeof(float),hipHostMallocMapped);
  hipHostAlloc(&distances_host,MAX_ITER*sizeof(float),hipHostMallocMapped);
  status = hipHostGetDevicePointer(&distances_cuda,distances_host,0);  
  status = hipHostGetDevicePointer(&new_vertex_cuda,new_vertex_host,0);  
  status = hipHostGetDevicePointer(&vertex_cuda_ptr,vertex_host_ptr,0);  
  hipEventCreateWithFlags(&stopEvent,hipEventDisableTiming);
  status = hipHostGetDevicePointer(&polygons_dev,polygons_,0);  
  double total_time = 0; 
  int loopcount = 0;
  int path_deadline_missed = 0;

	ofstream myfile;
	myfile.open ("RRT_time.txt");

//  //rabbitmq-c library//
//  //send channel
//  char const *hostname1;
//  int port1, astatus1;
//  char const *exchange1;
//  char const *routingkey1;
  char *s1;
//  amqp_socket_t *socket1 = NULL;
//  amqp_connection_state_t conn1;

//  hostname1 = "localhost";
//  port1 = 5672;
//  exchange1 = "amq.direct";
//  routingkey1 = "rrt_time";
////  messagebody1 = "hello world";

//  conn1 = amqp_new_connection();

//  socket1 = amqp_tcp_socket_new(conn1);
//  if (!socket1) {
//    die("creating TCP socket");
//  }

//  astatus1 = amqp_socket_open(socket1, hostname1, port1);
//  if (astatus1) {
//    die("opening TCP socket");
//  }

//  die_on_amqp_error(amqp_login(conn1, "/", 0, 131072, 0, AMQP_SASL_METHOD_PLAIN,
//                               "guest", "guest"),
//                    "Logging in");
//  amqp_channel_open(conn1, 1);
//  die_on_amqp_error(amqp_get_rpc_reply(conn1), "Opening channel");


//  //receive channel//////////////////////////////////////////
//  char const *hostname2;
//  int port2, astatus2;
//  char const *exchange2;
//  char const *bindingkey2;
//  //char *s2;
//  amqp_socket_t *socket = NULL;
//  amqp_connection_state_t conn2;
//  amqp_bytes_t queuename;

//  hostname2 = "localhost";
//  port2 = 5672;
//  exchange2 = "amq.direct";
//  bindingkey2 ="slamst";

//  conn2 = amqp_new_connection();

//  socket = amqp_tcp_socket_new(conn2);
//  if (!socket) {
//    die("creating TCP socket");
//  }

//  astatus2 = amqp_socket_open(socket, hostname2, port2);
//  if (astatus2) {
//    die("opening TCP socket");
//  }

//  die_on_amqp_error(amqp_login(conn2, "/", 0, 131072, 0, AMQP_SASL_METHOD_PLAIN,
//                               "guest", "guest"),
//                    "Logging in");
//  amqp_channel_open(conn2, 1);
//  die_on_amqp_error(amqp_get_rpc_reply(conn2), "Opening channel");

//  {
//    amqp_queue_declare_ok_t *r = amqp_queue_declare(
//        conn2, 1, amqp_empty_bytes, 0, 0, 0, 1, amqp_empty_table);
//    die_on_amqp_error(amqp_get_rpc_reply(conn2), "Declaring queue");
//    //queuename = amqp_bytes_malloc_dup(r->queue);
//    queuename = slamst;
//    if (queuename.bytes == NULL) {
//      fprintf(stderr, "Out of memory while copying queue name");
//      return 1;
//    }
//  }

//  amqp_queue_bind(conn2, 1, queuename, amqp_cstring_bytes(exchange2),
//                  amqp_cstring_bytes(bindingkey2), amqp_empty_table);
//  die_on_amqp_error(amqp_get_rpc_reply(conn2), "Binding queue");

//	  amqp_basic_consume(conn2, 1, queuename, amqp_empty_bytes, 0, 1, 0,
//		             amqp_empty_table);
//	  die_on_amqp_error(amqp_get_rpc_reply(conn2), "Consuming");

//	  {
//	    for (;;) {
//	      amqp_rpc_reply_t res;
//	      amqp_envelope_t envelope;

//	      amqp_maybe_release_buffers(conn2);

//	      res = amqp_consume_message(conn2, &envelope, NULL, 0);

//	      if (AMQP_RESPONSE_NORMAL != res.reply_type) {
//		break;
//	      }

//	      printf("Delivery %u, exchange2 %.*s routingkey %.*s\n",
//		     (unsigned)envelope.delivery_tag, (int)envelope.exchange.len,
//		     (char *)envelope.exchange.bytes, (int)envelope.routing_key.len,
//		     (char *)envelope.routing_key.bytes);

//	      if (envelope.message.properties._flags & AMQP_BASIC_CONTENT_TYPE_FLAG) {
//		printf("Content-type: %.*s\n",
//		       (int)envelope.message.properties.content_type.len,
//		       (char *)envelope.message.properties.content_type.bytes);
//	      }
//	      printf("----\n");

//	      amqp_dump(envelope.message.body.bytes, envelope.message.body.len);

//	      amqp_destroy_envelope(&envelope);
//	    }
//	  }

  ///////////////////////////////////////////////////////////

  for(loopcount = 0; loopcount < 10;loopcount++)
  {	
	  printf("inside for loop \n");


	  double begin_time = get_time_point();
  	myfile << begin_time ;
	sprintf(s1, "%lf",begin_time);

	  #ifdef WR2F
	  myfile1.open ("GPU_time.txt");
	  #endif
	  printf("GPU**===-------------------------------------------------===**\n");
	  rrt.exploreCuda();
	  #ifdef WR2F
	  myfile1.close();
	  #endif
	  double time_diff = ((double)get_time_point() - begin_time) / 1000;
  	myfile << time_diff ;
	  total_time+= time_diff;
	//  printf("%d :GPU path finding completed in %d loopcount \n",(loopcount+1),counter);
	  printf("Average Time for Path Planning is %lf milli-seconds \n", total_time/(loopcount+1));
	  int sleep_val = 2000.0-time_diff;
	  if(sleep_val > 0)
		usleep((sleep_val%1000)*1000);
	  	 else
	{
		printf("Path Planning Deadline missed \n");		
		path_deadline_missed++;
	}

//	rabbitmq-c process
//	  {
//	    amqp_basic_properties_t props;
//	    props._flags = AMQP_BASIC_CONTENT_TYPE_FLAG | AMQP_BASIC_DELIVERY_MODE_FLAG;
//	    props.content_type = amqp_cstring_bytes("text/plain");
//	    props.delivery_mode = 2; /* persistent delivery mode */
//	    die_on_error(amqp_basic_publish(conn1, 1, amqp_cstring_bytes(exchange1),
//		                            amqp_cstring_bytes(routingkey1), 0, 0,
//		                            &props, amqp_cstring_bytes(s1)),
//		         "Publishing");
//	  }

 	  //printf("GPU Path finding Completed in %lf milli-seconds.\n\n\n", ((double)get_time_point() - begin_time) / 1000);

	 /* begin_time = get_time_point();
	  #ifdef WR2F
	  myfile.open ("CPU_time.txt");
	  #endif
	  printf("CPU**===-------------------------------------------------===**\n");
	  rrt.exploreObstacles();
	  #ifdef WR2F
	  myfile.close();
	  #endif
	  printf("CPU Path finding Completed in %lf milli-seconds.\n\n\n", ((double)get_time_point() - begin_time) / 1000);*/
    }
    printf("Average Time for Path Planning is %lf milli-seconds \n", total_time/(loopcount));
   printf("Total Deadline Missed in Path Planning = %d\n",path_deadline_missed);
   //printf("Path Length = %d\n",vertex_count_);
//  die_on_amqp_error(amqp_channel_close(conn1, 1, AMQP_REPLY_SUCCESS),
//                    "Closing channel");
//  die_on_amqp_error(amqp_connection_close(conn1, AMQP_REPLY_SUCCESS),
//                    "Closing connection");
//  die_on_error(amqp_destroy_connection(conn1), "Ending connection");
	  myfile.close();
   destroy_cuda_mem();
   Destroy_stream();
return 0;

}









// end file

